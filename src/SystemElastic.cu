#include "hip/hip_runtime.h"
#include "include.cuh"
#include "System.cuh"

__device__ int ancf_shape_derivative_x(double* Sx, double x, double a)
{
  double xi = x/a;

  Sx[0] = (6*xi*xi-6*xi)/a;
  Sx[1] = 1-4*xi+3*xi*xi;
  Sx[2] = -(6*xi*xi-6*xi)/a;
  Sx[3] = -2*xi+3*xi*xi;

  return 0;
}

__device__ int ancf_shape_derivative2_x(double* Sxx, double x, double a)
{
  double xi = x/a;

  Sxx[0] = (12*xi-6)/(a*a);
  Sxx[1] = (-4+6*xi)/a;
  Sxx[2] = (6-12*xi)/(a*a);
  Sxx[3] = (-2+6*xi)/a;

  return 0;
}

__global__ void strainDerivativeUpdate(double ptj, double* p, double* strain, double* strainD, double* Sx, double3* geometries, int numBodies, int numBeams)
{
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numBeams);

  double a = geometries[index+numBodies].y;

  double x = .5*a*(1.+ptj);

  p = &p[12*index+3*numBodies];
  strainD = &strainD[12*index];
  Sx = &Sx[4*index];

  ancf_shape_derivative_x(Sx,x,a);

  strain[index] = .5*(((Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * p[0] + (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * p[1] + (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * p[2]) * Sx[0] + ((Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * p[3] + (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * p[4] + (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * p[5]) * Sx[1] + ((Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * p[6] + (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * p[7] + (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * p[8]) * Sx[2] + ((Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * p[9] + (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * p[10] + (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * p[11]) * Sx[3]-1);

  strainD[0]  = (Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * Sx[0];
  strainD[1]  = (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * Sx[0];
  strainD[2]  = (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * Sx[0];
  strainD[3]  = (Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * Sx[1];
  strainD[4]  = (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * Sx[1];
  strainD[5]  = (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * Sx[1];
  strainD[6]  = (Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * Sx[2];
  strainD[7]  = (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * Sx[2];
  strainD[8]  = (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * Sx[2];
  strainD[9]  = (Sx[0] * p[0] + Sx[1] * p[3] + Sx[2] * p[6] + Sx[3] * p[9]) * Sx[3];
  strainD[10] = (Sx[0] * p[1] + Sx[1] * p[4] + Sx[2] * p[7] + Sx[3] * p[10]) * Sx[3];
  strainD[11] = (Sx[0] * p[2] + Sx[1] * p[5] + Sx[2] * p[8] + Sx[3] * p[11]) * Sx[3];
}

__global__ void curvatDerivUpdate(double ptj, double* p, double* k, double* ke, double* Sx, double* Sxx, double3* geometries, int numBodies, int numBeams)
{
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numBeams);

  double a = geometries[index+numBodies].y;
  double x = .5*a*(1.+ptj);

  p = &p[12*index+3*numBodies];
  ke = &ke[12*index];
  Sx = &Sx[4*index];
  Sxx = &Sxx[4*index];

  ancf_shape_derivative_x(Sx,x,a);
  ancf_shape_derivative2_x(Sxx,x,a);

  double3 f1;
  double3 rx;
  double3 rxx;

  rx.x = p[0] * Sx[0] + p[3] * Sx[1] + p[6] * Sx[2] + p[9] * Sx[3];
  rx.y = p[1] * Sx[0] + p[4] * Sx[1] + p[7] * Sx[2] + p[10] * Sx[3];
  rx.z = p[2] * Sx[0] + p[5] * Sx[1] + p[8] * Sx[2] + p[11] * Sx[3];

  rxx.x = p[0] * Sxx[0] + p[3] * Sxx[1] + p[6] * Sxx[2] + p[9] * Sxx[3];
  rxx.y = p[1] * Sxx[0] + p[4] * Sxx[1] + p[7] * Sxx[2] + p[10] * Sxx[3];
  rxx.z = p[2] * Sxx[0] + p[5] * Sxx[1] + p[8] * Sxx[2] + p[11] * Sxx[3];

  double g1 = sqrt(rx.x*rx.x+rx.y*rx.y+rx.z*rx.z);
  double g = pow(g1,3);

  f1.x = rx.y * rxx.z - rx.z * rxx.y;
  f1.y = rx.z * rxx.x - rx.x * rxx.z;
  f1.z = rx.x * rxx.y - rx.y * rxx.x;

  double f = sqrt(f1.x*f1.x+f1.y*f1.y+f1.z*f1.z);

  k[index] = f/g;

  double fspecial = -1.0;
  if(f) fspecial = fspecial/f;

  ke[0] = pow(g, -0.2e1) * (g * (fspecial * f1.y * (-Sx[0] * rxx.z - rx.z * Sxx[0]) + fspecial * f1.z * (Sx[0] * rxx.y + rx.y * Sxx[0])) - f * (0.3e1 * g1 * Sx[0] * p[0] + 0.3e1 * g1 * Sx[1] * p[3] + 0.3e1 * g1 * Sx[2] * p[6] + 0.3e1 * g1 * Sx[3] * p[9]) * Sx[0]);
  ke[1] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (Sx[0] * rxx.z + rx.z * Sxx[0]) + fspecial * f1.z * (-Sx[0] * rxx.x - rx.x * Sxx[0])) - f * (0.3e1 * g1 * Sx[0] * p[1] + 0.3e1 * g1 * Sx[1] * p[4] + 0.3e1 * g1 * Sx[2] * p[7] + 0.3e1 * g1 * Sx[3] * p[10]) * Sx[0]);
  ke[2] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (-Sx[0] * rxx.y - rx.y * Sxx[0]) + fspecial * f1.y * (Sx[0] * rxx.x + rx.x * Sxx[0])) - f * (0.3e1 * g1 * Sx[0] * p[2] + 0.3e1 * g1 * Sx[1] * p[5] + 0.3e1 * g1 * Sx[2] * p[8] + 0.3e1 * g1 * Sx[3] * p[11]) * Sx[0]);
  ke[3] = pow(g, -0.2e1) * (g * (fspecial * f1.y * (-Sx[1] * rxx.z - rx.z * Sxx[1]) + fspecial * f1.z * (Sx[1] * rxx.y + rx.y * Sxx[1])) - f * (0.3e1 * g1 * Sx[0] * p[0] + 0.3e1 * g1 * Sx[1] * p[3] + 0.3e1 * g1 * Sx[2] * p[6] + 0.3e1 * g1 * Sx[3] * p[9]) * Sx[1]);
  ke[4] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (Sx[1] * rxx.z + rx.z * Sxx[1]) + fspecial * f1.z * (-Sx[1] * rxx.x - rx.x * Sxx[1])) - f * (0.3e1 * g1 * Sx[0] * p[1] + 0.3e1 * g1 * Sx[1] * p[4] + 0.3e1 * g1 * Sx[2] * p[7] + 0.3e1 * g1 * Sx[3] * p[10]) * Sx[1]);
  ke[5] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (-Sx[1] * rxx.y - rx.y * Sxx[1]) + fspecial * f1.y * (Sx[1] * rxx.x + rx.x * Sxx[1])) - f * (0.3e1 * g1 * Sx[0] * p[2] + 0.3e1 * g1 * Sx[1] * p[5] + 0.3e1 * g1 * Sx[2] * p[8] + 0.3e1 * g1 * Sx[3] * p[11]) * Sx[1]);
  ke[6] = pow(g, -0.2e1) * (g * (fspecial * f1.y * (-Sx[2] * rxx.z - rx.z * Sxx[2]) + fspecial * f1.z * (Sx[2] * rxx.y + rx.y * Sxx[2])) - f * (0.3e1 * g1 * Sx[0] * p[0] + 0.3e1 * g1 * Sx[1] * p[3] + 0.3e1 * g1 * Sx[2] * p[6] + 0.3e1 * g1 * Sx[3] * p[9]) * Sx[2]);
  ke[7] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (Sx[2] * rxx.z + rx.z * Sxx[2]) + fspecial * f1.z * (-Sx[2] * rxx.x - rx.x * Sxx[2])) - f * (0.3e1 * g1 * Sx[0] * p[1] + 0.3e1 * g1 * Sx[1] * p[4] + 0.3e1 * g1 * Sx[2] * p[7] + 0.3e1 * g1 * Sx[3] * p[10]) * Sx[2]);
  ke[8] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (-Sx[2] * rxx.y - rx.y * Sxx[2]) + fspecial * f1.y * (Sx[2] * rxx.x + rx.x * Sxx[2])) - f * (0.3e1 * g1 * Sx[0] * p[2] + 0.3e1 * g1 * Sx[1] * p[5] + 0.3e1 * g1 * Sx[2] * p[8] + 0.3e1 * g1 * Sx[3] * p[11]) * Sx[2]);
  ke[9] = pow(g, -0.2e1) * (g * (fspecial * f1.y * (-Sx[3] * rxx.z - rx.z * Sxx[3]) + fspecial * f1.z * (Sx[3] * rxx.y + rx.y * Sxx[3])) - f * (0.3e1 * g1 * Sx[0] * p[0] + 0.3e1 * g1 * Sx[1] * p[3] + 0.3e1 * g1 * Sx[2] * p[6] + 0.3e1 * g1 * Sx[3] * p[9]) * Sx[3]);
  ke[10] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (Sx[3] * rxx.z + rx.z * Sxx[3]) + fspecial * f1.z * (-Sx[3] * rxx.x - rx.x * Sxx[3])) - f * (0.3e1 * g1 * Sx[0] * p[1] + 0.3e1 * g1 * Sx[1] * p[4] + 0.3e1 * g1 * Sx[2] * p[7] + 0.3e1 * g1 * Sx[3] * p[10]) * Sx[3]);
  ke[11] = pow(g, -0.2e1) * (g * (fspecial * f1.x * (-Sx[3] * rxx.y - rx.y * Sxx[3]) + fspecial * f1.y * (Sx[3] * rxx.x + rx.x * Sxx[3])) - f * (0.3e1 * g1 * Sx[0] * p[2] + 0.3e1 * g1 * Sx[1] * p[5] + 0.3e1 * g1 * Sx[2] * p[8] + 0.3e1 * g1 * Sx[3] * p[11]) * Sx[3]);
}

__global__ void addInternalForceComponent(double* f, double* strainD_shared, double* strainVec, double3* materials, double3* geometries, double wtl, int numBodies, int numBeams, int check)
{
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numBeams);

  double strain = strainVec[index];
  double3 geometry = geometries[index+numBodies];
  double E = materials[index].y;
  double a = geometry.y;
  double r = geometry.x;
  double A = PI*r*r;
  double I = .25*PI*r*r*r*r;

  f = &f[12*index+3*numBodies];
  strainD_shared = &strainD_shared[12*index];
  double factor = wtl*A*E*a*.5;
  if(check) factor = wtl*I*E*a*.5;

  f[0] += factor * strain * strainD_shared[0];
  f[1] += factor * strain * strainD_shared[1];
  f[2] += factor * strain * strainD_shared[2];
  f[3] += factor * strain * strainD_shared[3];
  f[4] += factor * strain * strainD_shared[4];
  f[5] += factor * strain * strainD_shared[5];
  f[6] += factor * strain * strainD_shared[6];
  f[7] += factor * strain * strainD_shared[7];
  f[8] += factor * strain * strainD_shared[8];
  f[9] += factor * strain * strainD_shared[9];
  f[10] += factor * strain * strainD_shared[10];
  f[11] += factor * strain * strainD_shared[11];
}

int System::updateElasticForces()
{
  thrust::fill(fElastic_d.begin(),fElastic_d.end(),0.0); //Clear internal forces

  for(int j=0;j<pt5.size();j++)
  {
    strainDerivativeUpdate<<<BLOCKS(beams.size()),THREADS>>>(pt5[j],CASTD1(p_d),CASTD1(strain_d),CASTD1(strainDerivative_d),CASTD1(Sx_d),CASTD3(contactGeometry_d),bodies.size(),beams.size());
    addInternalForceComponent<<<BLOCKS(beams.size()),THREADS>>>(CASTD1(fElastic_d),CASTD1(strainDerivative_d),CASTD1(strain_d),CASTD3(materialsBeam_d),CASTD3(contactGeometry_d),wt5[j],bodies.size(),beams.size(),0);
  }

  for(int j=0;j<pt3.size();j++)
  {
    curvatDerivUpdate<<<BLOCKS(beams.size()),THREADS>>>(pt3[j],CASTD1(p_d),CASTD1(strain_d),CASTD1(strainDerivative_d),CASTD1(Sx_d),CASTD1(Sxx_d),CASTD3(contactGeometry_d),bodies.size(),beams.size());
    addInternalForceComponent<<<BLOCKS(beams.size()),THREADS>>>(CASTD1(fElastic_d),CASTD1(strainDerivative_d),CASTD1(strain_d),CASTD3(materialsBeam_d),CASTD3(contactGeometry_d),wt3[j],bodies.size(),beams.size(),1);
  }

  return 0;
}
