#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "System.cuh"

System::System()
{
  gravity = make_double3(0,-9.81,0);
  tol = 1e-8;
  h = 1e-3;
  timeIndex = 0;
  time = 0;

	// spike stuff
	partitions = 1;
	solverOptions.safeFactorization = true;
	solverOptions.trackReordering = true;
	solverOptions.maxNumIterations = 5000;
	preconditionerUpdateModulus = -1; // the preconditioner updates every ___ time steps
	preconditionerMaxKrylovIterations = -1; // the preconditioner updates if Krylov iterations are greater than ____ iterations
	//mySolver = new SpikeSolver(partitions, solverOptions);
	//m_spmv = new MySpmv(mass);
  stepKrylovIterations = 0;
  precUpdated = 0;
	// end spike stuff

  collisionDetector = new CollisionDetector(this);
}

void System::setSolverType(int solverType)
{
	switch(solverType) {
	case 0:
		solverOptions.solverType = spike::BiCGStab;
		break;
	case 1:
		solverOptions.solverType = spike::BiCGStab1;
		break;
	case 2:
		solverOptions.solverType = spike::BiCGStab2;
		break;
	case 3:
		solverOptions.solverType = spike::MINRES;
		break;
	}
}

void System::setPrecondType(int useSpike)
{
	solverOptions.precondType = useSpike ? spike::Spike : spike::None;
}

void System::setTimeStep(double step_size, double precision)
{
	h = step_size;

	// Set tolerance for Newton iteration based on the precision in positions
	// and integration step-size.
	double safety = 1;////0.5;
	tol = safety * precision / (h * h);

	// Set the tolerances for Krylov
	solverOptions.relTol = std::min(0.01 * tol, 1e-6);
	solverOptions.absTol = 1e-10;
}

void System::printSolverParams()
{
	printf("Step size: %e\n", h);
	printf("Newton tolerance: %e\n", tol);
	printf("Krylov relTol: %e  abdTol: %e\n", solverOptions.relTol, solverOptions.absTol);
	printf("Max. Krylov iterations: %d\n", solverOptions.maxNumIterations);
	printf("----------------------------\n");
}

int System::add(Body* body) {
  // TODO: make this function general for any Body
	//add the element
  body->setIndex(p_h.size()); // Indicates the Body's location in the position array
  indices_h.push_back(p_h.size()); // Push Body's location to global library
  body->setIdentifier(bodies.size()); // Indicates the number that the Body was added
	bodies.push_back(body);

	// update p
	p_h.push_back(body->pos.x);
	p_h.push_back(body->pos.y);
	p_h.push_back(body->pos.z);

  // update v
  v_h.push_back(body->vel.x);
  v_h.push_back(body->vel.y);
  v_h.push_back(body->vel.z);

  // update a
  a_h.push_back(body->acc.x);
  a_h.push_back(body->acc.y);
  a_h.push_back(body->acc.z);

	// update external force vector (gravity)
  if(body->isFixed()) {
    f_h.push_back(0);
    f_h.push_back(0);
    f_h.push_back(0);
  }
  else {
    f_h.push_back(body->mass * this->gravity.x);
    f_h.push_back(body->mass * this->gravity.y);
    f_h.push_back(body->mass * this->gravity.z);
  }

  f_contact_h.push_back(0);
  f_contact_h.push_back(0);
  f_contact_h.push_back(0);

  tmp_h.push_back(0);
  tmp_h.push_back(0);
  tmp_h.push_back(0);

  r_h.push_back(0);
  r_h.push_back(0);
  r_h.push_back(0);

  k_h.push_back(0);
  k_h.push_back(0);
  k_h.push_back(0);

  for(int i=0; i<3; i++) {
    gamma_h.push_back(0);
    gammaHat_h.push_back(0);
    gammaNew_h.push_back(0);
    g_h.push_back(0);
    y_h.push_back(0);
    yNew_h.push_back(0);
    gammaTmp_h.push_back(0);
  }

	// update the mass matrix
	for (int i = 0; i < body->numDOF; i++) {
	  //if(!body->isFixed()) {
      massI_h.push_back(i + body->numDOF * (bodies.size() - 1));
      massJ_h.push_back(i + body->numDOF * (bodies.size() - 1));
      mass_h.push_back(1.0/body->mass);
	  //}
	}

	contactGeometry_h.push_back(body->contactGeometry);

	return bodies.size();
}

int System::initializeDevice() {
  indices_d = indices_h;
	p_d = p_h;
	v_d = v_h;
	a_d = a_h;
	f_d = f_h;
	f_contact_d = f_contact_h;
	tmp_d = tmp_h;
	r_d = r_h;
	k_d = k_h;
  gamma_d = gamma_h;
  gammaHat_d = gammaHat_h;
  gammaNew_d = gammaNew_h;
  g_d = g_h;
  y_d = y_h;
  yNew_d = yNew_h;
  gammaTmp_d = gammaTmp_h;

	massI_d = massI_h;
	massJ_d = massJ_h;
	mass_d = mass_h;

	contactGeometry_d = contactGeometry_h;
	fixedBodies_d = fixedBodies_h;

	thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
	thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
	thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
	thrust::device_ptr<double> wrapped_device_f(CASTD1(f_d));
	thrust::device_ptr<double> wrapped_device_f_contact(CASTD1(f_contact_d));
	thrust::device_ptr<double> wrapped_device_tmp(CASTD1(tmp_d));
	thrust::device_ptr<double> wrapped_device_r(CASTD1(r_d));
	thrust::device_ptr<double> wrapped_device_k(CASTD1(k_d));
	thrust::device_ptr<double> wrapped_device_gamma(CASTD1(gamma_d));
	thrust::device_ptr<double> wrapped_device_gammaHat(CASTD1(gammaHat_d));
	thrust::device_ptr<double> wrapped_device_gammaNew(CASTD1(gammaNew_d));
	thrust::device_ptr<double> wrapped_device_g(CASTD1(g_d));
	thrust::device_ptr<double> wrapped_device_y(CASTD1(y_d));
	thrust::device_ptr<double> wrapped_device_yNew(CASTD1(yNew_d));
	thrust::device_ptr<double> wrapped_device_gammaTmp(CASTD1(gammaTmp_d));

	p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
	v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
	a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
	f = DeviceValueArrayView(wrapped_device_f, wrapped_device_f + f_d.size());
	f_contact = DeviceValueArrayView(wrapped_device_f_contact, wrapped_device_f_contact + f_contact_d.size());
	tmp = DeviceValueArrayView(wrapped_device_tmp, wrapped_device_tmp + tmp_d.size());
	r = DeviceValueArrayView(wrapped_device_r, wrapped_device_r + r_d.size());
	k = DeviceValueArrayView(wrapped_device_k, wrapped_device_k + k_d.size());
	gamma = DeviceValueArrayView(wrapped_device_gamma, wrapped_device_gamma + gamma_d.size());
	gammaHat = DeviceValueArrayView(wrapped_device_gammaHat, wrapped_device_gammaHat + gammaHat_d.size());
	gammaNew = DeviceValueArrayView(wrapped_device_gammaNew, wrapped_device_gammaNew + gammaNew_d.size());
	g = DeviceValueArrayView(wrapped_device_g, wrapped_device_g + g_d.size());
	y = DeviceValueArrayView(wrapped_device_y, wrapped_device_y + y_d.size());
	yNew = DeviceValueArrayView(wrapped_device_yNew, wrapped_device_yNew + yNew_d.size());
	gammaTmp = DeviceValueArrayView(wrapped_device_gammaTmp, wrapped_device_gammaTmp + gammaTmp_d.size());

	// create mass matrix using cusp library (shouldn't change)
	thrust::device_ptr<int> wrapped_device_I(CASTI1(massI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + massI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(massJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + massJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(mass_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + mass_d.size());

	mass = DeviceView(a_d.size(), a_d.size(), mass_d.size(), row_indices, column_indices, values);
	// end create mass matrix

	return 0;
}

int System::initializeSystem() {

  // update the contact geometry and fixed bodies
  for(int i=0; i<bodies.size(); i++) {
    contactGeometry_h[i] = bodies[i]->contactGeometry;
    if(bodies[i]->isFixed()) fixedBodies_h.push_back(i);
  }

	initializeDevice();

	// create and setup the Spike::GPU solver
	//m_spmv = new MySpmv(mass);
	//mySolver = new SpikeSolver(partitions, solverOptions);
	//mySolver->setup(mass);

	//bool success = mySolver->solve(*m_spmv, f, a);

	//collisionDetector->detectPossibleCollisions_nSquared();

	return 0;
}

int System::DoTimeStep() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Perform collision detection
	collisionDetector->generateAxisAlignedBoundingBoxes();
	collisionDetector->detectPossibleCollisions_spatialSubdivision();
  collisionDetector->detectCollisions();

  // Set up the QOCC
  buildContactJacobian();
  buildRightHandSideVector();

  // Solve the QOCC



  cusp::blas::axpy(f, f_contact, 1.0);

  fixBodies();

	cusp::multiply(mass, f_contact, a);
	//bool success = mySolver->solve(*m_spmv, f, a);
	cusp::blas::axpy(a, v, h);
	cusp::blas::axpy(v, p, h);

  time += h;
  timeIndex++;
  p_h = p_d;

  printf("Time: %f, Collisions: %d (%d possible)\n",time,collisionDetector->numCollisions, (int)collisionDetector->numPossibleCollisions);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	return 0;
}

__global__ void addContactForces(double* f, uint* collisionStartIndex, int* indices, double* v, double4* normalsAndPenetrations, uint* bodyIdentifiersA, uint* bodyIdentifiersB, uint lastActiveCollision) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, lastActiveCollision);

  int bodyA = bodyIdentifiersA[index];
  int bodyIndexA = indices[bodyA];
  double3 velA = make_double3(v[bodyIndexA],v[bodyIndexA+1],v[bodyIndexA+2]);
  uint startIndex = (index == 0) ? 0 : collisionStartIndex[index - 1];
  uint endIndex = collisionStartIndex[index];

  double3 force = make_double3(0,0,0);
  double3 normal = make_double3(0,0,0);
  double penetration = 0;
  double4 normalAndPenetration = make_double4(0,0,0,0);
  for (int i = startIndex; i < endIndex; i++) {
    // TODO: Replace with actual material/geometry
    double sigmaA = (1.0-0.25)/2.0e7;
    double sigmaB = sigmaA;
    double rA = 0.4;
    double rB = 0.4;
    normalAndPenetration = normalsAndPenetrations[i];
    penetration = normalAndPenetration.w;
    normal = make_double3(normalAndPenetration.x,normalAndPenetration.y,normalAndPenetration.z);

    force += 4.0/(3.0*(sigmaA+sigmaB))*sqrt(rA*rB/(rA+rB))*pow(penetration,1.5)*normal;

    // Add damping
    int bodyB = bodyIdentifiersB[i];
    int bodyIndexB = indices[bodyB];
    double3 velB = make_double3(v[bodyIndexB],v[bodyIndexB+1],v[bodyIndexB+2]);
    double3 vel = velB-velA;
    double b = 250; //TODO: Add to material library
    double3 damping;
    damping.x = b * normal.x * normal.x * vel.x + b * normal.x * normal.y * vel.y + b * normal.x * normal.z * vel.z;
    damping.y = b * normal.x * normal.y * vel.x + b * normal.y * normal.y * vel.y + b * normal.y * normal.z * vel.z;
    damping.z = b * normal.x * normal.z * vel.x + b * normal.y * normal.z * vel.y + b * normal.z * normal.z * vel.z;
    if(penetration>0) force += damping;
  }

  f[bodyIndexA]   += force.x;
  f[bodyIndexA+1] += force.y;
  f[bodyIndexA+2] += force.z;
}

int System::applyContactForces() {
  Thrust_Fill(f_contact_d,0);
  if(collisionDetector->numCollisions) {
    addContactForces<<<BLOCKS(collisionDetector->lastActiveCollision),THREADS>>>(CASTD1(f_contact_d), CASTU1(collisionDetector->collisionStartIndex_d), CASTI1(indices_d), CASTD1(v_d), CASTD4(collisionDetector->normalsAndPenetrations_d), CASTU1(collisionDetector->bodyIdentifierA_d), CASTU1(collisionDetector->bodyIdentifierB_d), collisionDetector->lastActiveCollision);
  }

  return 0;
}

int System::applyContactForces_CPU() {
  Thrust_Fill(f_contact_h,0);

  for(int i=0; i<collisionDetector->normalsAndPenetrations_h.size(); i++) {
    uint bodyA = collisionDetector->bodyIdentifierA_h[i];
    uint bodyB = collisionDetector->bodyIdentifierB_h[i];
    double4 nAndP = collisionDetector->normalsAndPenetrations_h[i];
    double3 normal = make_double3(nAndP.x,nAndP.y,nAndP.z);
    double penetration = nAndP.w;

    double sigmaA = (1.0-0.25)/2.0e7;
    double sigmaB = sigmaA;
    double rA = 0.4;
    double rB = 0.4;
    double3 contactForce = 4.0/(3.0*(sigmaA+sigmaB))*sqrt(rA*rB/(rA+rB))*pow(penetration,1.5)*normal;

    // Add damping
    v_h = v_d;
    double3 v = make_double3(v_h[indices_h[bodyB]]-v_h[indices_h[bodyA]],v_h[indices_h[bodyB]+1]-v_h[indices_h[bodyA]+1],v_h[indices_h[bodyB]+2]-v_h[indices_h[bodyA]+2]);
    double b = 250;
    double3 damping;
    damping.x = b * normal.x * normal.x * v.x + b * normal.x * normal.y * v.y + b * normal.x * normal.z * v.z;
    damping.y = b * normal.x * normal.y * v.x + b * normal.y * normal.y * v.y + b * normal.y * normal.z * v.z;
    damping.z = b * normal.x * normal.z * v.x + b * normal.y * normal.z * v.y + b * normal.z * normal.z * v.z;
    if(penetration>=0) contactForce -= damping;

    f_contact_h[indices_h[bodyA]]   -= contactForce.x;
    f_contact_h[indices_h[bodyA]+1] -= contactForce.y;
    f_contact_h[indices_h[bodyA]+2] -= contactForce.z;

    f_contact_h[indices_h[bodyB]]   += contactForce.x;
    f_contact_h[indices_h[bodyB]+1] += contactForce.y;
    f_contact_h[indices_h[bodyB]+2] += contactForce.z;

  }
  f_contact_d = f_contact_h;

  return 0;
}

__global__ void fixFixedBodies(double* f, int* indices, int* fixedBodies, uint numFixedBodies) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numFixedBodies);

  int body = fixedBodies[index];
  int bodyIndex = indices[body];

  f[bodyIndex]   = 0;
  f[bodyIndex+1] = 0;
  f[bodyIndex+2] = 0;
}

int System::fixBodies() {
  if(fixedBodies_d.size()) {
    fixFixedBodies<<<BLOCKS(fixedBodies_d.size()),THREADS>>>(CASTD1(f_contact_d), CASTI1(indices_d), CASTI1(fixedBodies_d), fixedBodies_d.size());
  }

  return 0;
}

int System::buildContactJacobian() {
  if(collisionDetector->numCollisions) {
    // TODO: Perform this in parallel!
    DI_h.clear();
    DJ_h.clear();
    D_h.clear();
    double4 nAndP;
    double3 n, u, v;
    uint bodyA, bodyB;
    for(int i=0; i<collisionDetector->numCollisions; i++) {
      bodyA = collisionDetector->bodyIdentifierA_h[i];
      bodyB = collisionDetector->bodyIdentifierB_h[i];
      nAndP = collisionDetector->normalsAndPenetrations_h[i];
      n = make_double3(nAndP.x,nAndP.y,nAndP.z);

      if(n.z != 0) {
        u = normalize(make_double3(1,0,-n.x/n.z));
      }
      else if(n.x != 0) {
        u = normalize(make_double3(-n.z/n.x,0,1));
      }
      else {
        u = normalize(make_double3(1,-n.x/n.y,0));
      }
      v = normalize(cross(n,u));

      DI_h.push_back(3*i+0);
      DI_h.push_back(3*i+0);
      DI_h.push_back(3*i+0);
      DI_h.push_back(3*i+0);
      DI_h.push_back(3*i+0);
      DI_h.push_back(3*i+0);

      DJ_h.push_back(indices_h[bodyA]+0);
      DJ_h.push_back(indices_h[bodyA]+1);
      DJ_h.push_back(indices_h[bodyA]+2);
      DJ_h.push_back(indices_h[bodyB]+0);
      DJ_h.push_back(indices_h[bodyB]+1);
      DJ_h.push_back(indices_h[bodyB]+2);

      D_h.push_back(-n.x);
      D_h.push_back(-n.y);
      D_h.push_back(-n.z);
      D_h.push_back(n.x);
      D_h.push_back(n.y);
      D_h.push_back(n.z);

      DI_h.push_back(3*i+1);
      DI_h.push_back(3*i+1);
      DI_h.push_back(3*i+1);
      DI_h.push_back(3*i+1);
      DI_h.push_back(3*i+1);
      DI_h.push_back(3*i+1);

      DJ_h.push_back(indices_h[bodyA]+0);
      DJ_h.push_back(indices_h[bodyA]+1);
      DJ_h.push_back(indices_h[bodyA]+2);
      DJ_h.push_back(indices_h[bodyB]+0);
      DJ_h.push_back(indices_h[bodyB]+1);
      DJ_h.push_back(indices_h[bodyB]+2);

      D_h.push_back(-u.x);
      D_h.push_back(-u.y);
      D_h.push_back(-u.z);
      D_h.push_back(u.x);
      D_h.push_back(u.y);
      D_h.push_back(u.z);

      DI_h.push_back(3*i+2);
      DI_h.push_back(3*i+2);
      DI_h.push_back(3*i+2);
      DI_h.push_back(3*i+2);
      DI_h.push_back(3*i+2);
      DI_h.push_back(3*i+2);

      DJ_h.push_back(indices_h[bodyA]+0);
      DJ_h.push_back(indices_h[bodyA]+1);
      DJ_h.push_back(indices_h[bodyA]+2);
      DJ_h.push_back(indices_h[bodyB]+0);
      DJ_h.push_back(indices_h[bodyB]+1);
      DJ_h.push_back(indices_h[bodyB]+2);

      D_h.push_back(-v.x);
      D_h.push_back(-v.y);
      D_h.push_back(-v.z);
      D_h.push_back(v.x);
      D_h.push_back(v.y);
      D_h.push_back(v.z);
    }

    DI_d = DI_h;
    DJ_d = DJ_h;
    D_d = D_h;

    DTI_d = DI_d;
    DTJ_d = DJ_d;
    DT_d = D_d;

    // create contact jacobian using cusp library
    thrust::device_ptr<int> wrapped_device_I(CASTI1(DI_d));
    DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + DI_d.size());

    thrust::device_ptr<int> wrapped_device_J(CASTI1(DJ_d));
    DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + DJ_d.size());

    thrust::device_ptr<double> wrapped_device_V(CASTD1(D_d));
    DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + D_d.size());

    D = DeviceView(3*collisionDetector->numCollisions, 3*bodies.size(), D_d.size(), row_indices, column_indices, values);
    // end create contact jacobian

    buildContactJacobianTranspose();
  }

  return 0;
}

int System::buildContactJacobianTranspose() {
  DTI_d = DJ_d;
  DTJ_d = DI_d;
  DT_d = D_d;

  // create contact jacobian using cusp library
  thrust::device_ptr<int> wrapped_device_I(CASTI1(DTI_d));
  DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + DI_d.size());

  thrust::device_ptr<int> wrapped_device_J(CASTI1(DTJ_d));
  DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + DJ_d.size());

  thrust::device_ptr<double> wrapped_device_V(CASTD1(DT_d));
  DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + D_d.size());

  DT = DeviceView(3*bodies.size(), 3*collisionDetector->numCollisions, DT_d.size(), row_indices, column_indices, values);
  // end create contact jacobian

  DT.sort_by_row(); // TODO: Do I need this?

  return 0;
}

int System::performSchurComplementProduct(DeviceValueArrayView src, DeviceValueArrayView dst) {
  cusp::multiply(DT,src,tmp);
  cusp::multiply(mass,tmp,tmp);
  cusp::multiply(D,tmp,dst);

  return 0;
}

__global__ void applyStabilization(double* r, double4* normalsAndPenetrations, double timeStep, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double penetration = normalsAndPenetrations[index].w;

  r[3*index] += penetration/timeStep;
}

int System::buildRightHandSideVector() {
  // build k
  cusp::multiply(mass,v,k);
  cusp::blas::axpy(f,k,h);

  // build r
  r_d.resize(3*collisionDetector->numCollisions);
  r.resize(3*collisionDetector->numCollisions);
  cusp::multiply(mass,k,tmp);
  cusp::multiply(DT,tmp,r);
  applyStabilization<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTD1(r_d), CASTD4(collisionDetector->normalsAndPenetrations_d), h, collisionDetector->numCollisions);

  return 0;
}

int System::project(thrust::device_vector<double> src) {

  return 0;
}

int System::solve_APGD() {
  int maxIterations = 500;
  double tolerance = 1e-3;

  gamma_d.resize(3*collisionDetector->numCollisions);
  gammaHat_d.resize(3*collisionDetector->numCollisions);
  gammaNew_d.resize(3*collisionDetector->numCollisions);
  g_d.resize(3*collisionDetector->numCollisions);
  y_d.resize(3*collisionDetector->numCollisions);
  yNew_d.resize(3*collisionDetector->numCollisions);
  gammaTmp_d.resize(3*collisionDetector->numCollisions);
  gamma.resize(3*collisionDetector->numCollisions);
  gammaHat.resize(3*collisionDetector->numCollisions);
  gammaNew.resize(3*collisionDetector->numCollisions);
  g.resize(3*collisionDetector->numCollisions);
  y.resize(3*collisionDetector->numCollisions);
  yNew.resize(3*collisionDetector->numCollisions);
  gammaTmp.resize(3*collisionDetector->numCollisions);

  // (1) gamma_0 = zeros(nc,1)
  cusp::blas::fill(gamma,0);

  // (2) gamma_hat_0 = ones(nc,1)
  cusp::blas::fill(gammaHat,1.0);

  // (3) y_0 = gamma_0
  cusp::blas::copy(gamma,y);

  // (4) theta_0 = 1
  double theta = 1.0;
  double thetaNew = theta;
  double Beta = 0.0;
  double obj1 = 0.0;
  double obj2 = 0.0;
  double residual = 10e30;

  // (5) L_k = norm(N * (gamma_0 - gamma_hat_0)) / norm(gamma_0 - gamma_hat_0)
  cusp::blas::axpby(gamma,gammaHat,gammaTmp,1.0,-1.0);
  double L = cusp::blas::nrm2(gammaTmp);
  performSchurComplementProduct(gammaTmp, gammaTmp);
  L = cusp::blas::nrm2(gammaTmp)/L;

  // (6) t_k = 1 / L_k
  double t = 1.0/L;

  // (7) for k := 0 to N_max
  for (int k = 0; k < maxIterations; k++) {
    // (8) g = N * y_k - r
    performSchurComplementProduct(y, g);
    cusp::blas::axpy(r,g,-1.0);

    // (9) gamma_(k+1) = ProjectionOperator(y_k - t_k * g)
    cusp::blas::axpby(y,g,gammaNew,1.0,-t);
    project(gammaNew_d);
  }

  return 0;
}
