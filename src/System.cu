#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "System.cuh"
#include "Solver.cuh"
#include "APGD.cuh"
#include "PDIP.cuh"
#include "TPAS.cuh"
#include "JKIP.cuh"
#include "PJKIP.cuh"
#include "PGJ.cuh"
#include "PGS.cuh"

System::System()
{
  gravity = make_double3(0,-9.81,0);
  tol = 1e-8;
  h = 1e-3;
  timeIndex = 0;
  time = 0;
  elapsedTime = 0;
  totalGPUMemoryUsed = 0;

  collisionDetector = new CollisionDetector(this);
  solver = new APGD(this);

  wt3.push_back(5.0 / 9.0);
  wt3.push_back(8.0 / 9.0);
  wt3.push_back(5.0 / 9.0);
  pt3.push_back(-sqrt(3.0 / 5.0));
  pt3.push_back(0.0);
  pt3.push_back(sqrt(3.0 / 5.0));

  wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
  wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
  wt5.push_back(128. / 225.);
  wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
  wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
  pt5.push_back(-(sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back(-(sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back(0.);
  pt5.push_back((sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back((sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
}

System::System(int solverType)
{
  gravity = make_double3(0,-9.81,0);
  tol = 1e-8;
  h = 1e-3;
  timeIndex = 0;
  time = 0;
  elapsedTime = 0;
  totalGPUMemoryUsed = 0;

  collisionDetector = new CollisionDetector(this);

  switch(solverType) {
  case 1:
    solver = new APGD(this);
    break;
  case 2:
    solver = new PDIP(this);
    break;
  case 3:
    solver = new TPAS(this);
    break;
  case 4:
    solver = new JKIP(this);
    break;
  case 5:
    solver = new PGJ(this);
    break;
  case 6:
    solver = new PGS(this);
    break;
  case 7:
    solver = new PJKIP(this);
    break;
  default:
    solver = new APGD(this);
  }

  wt3.push_back(5.0 / 9.0);
  wt3.push_back(8.0 / 9.0);
  wt3.push_back(5.0 / 9.0);
  pt3.push_back(-sqrt(3.0 / 5.0));
  pt3.push_back(0.0);
  pt3.push_back(sqrt(3.0 / 5.0));

  wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
  wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
  wt5.push_back(128. / 225.);
  wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
  wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
  pt5.push_back(-(sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back(-(sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back(0.);
  pt5.push_back((sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
  pt5.push_back((sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
}

void System::setTimeStep(double step_size)
{
  h = step_size;
}

int System::add(Body* body) {
  //add the body
  bodies.push_back(body);

  return bodies.size();
}

int System::add(Beam* beam) {
  //add the beam
  beam->sys = this;
  beams.push_back(beam);
  return beams.size();
}

int System::initializeDevice() {

  indices_d = indices_h;
  p_d = p_h;
  v_d = v_h;
  a_d = a_h;
  f_d = f_h;
  f_contact_d = f_contact_h;
  tmp_d = tmp_h;
  r_d = r_h;
  b_d = b_h;
  k_d = k_h;
  gamma_d = a_h;
  friction_d = a_h;
  fApplied_d = fApplied_h;
  fElastic_d = fElastic_h;

  massI_d = massI_h;
  massJ_d = massJ_h;
  mass_d = mass_h;

  contactGeometry_d = contactGeometry_h;
  collisionGeometry_d = collisionGeometry_h;
  collisionMap_d = collisionMap_h;
  materialsBeam_d = materialsBeam_h;
  fixedBodies_d = fixedBodies_h;

  strainDerivative_d = strainDerivative_h;
  strain_d = strain_h;
  Sx_d = Sx_h;
  Sxx_d = Sxx_h;

  thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
  thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
  thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
  thrust::device_ptr<double> wrapped_device_f(CASTD1(f_d));
  thrust::device_ptr<double> wrapped_device_f_contact(CASTD1(f_contact_d));
  thrust::device_ptr<double> wrapped_device_fApplied(CASTD1(fApplied_d));
  thrust::device_ptr<double> wrapped_device_fElastic(CASTD1(fElastic_d));
  thrust::device_ptr<double> wrapped_device_tmp(CASTD1(tmp_d));
  thrust::device_ptr<double> wrapped_device_r(CASTD1(r_d));
  thrust::device_ptr<double> wrapped_device_b(CASTD1(b_d));
  thrust::device_ptr<double> wrapped_device_k(CASTD1(k_d));
  thrust::device_ptr<double> wrapped_device_gamma(CASTD1(gamma_d));

  p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
  v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
  a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
  f = DeviceValueArrayView(wrapped_device_f, wrapped_device_f + f_d.size());
  f_contact = DeviceValueArrayView(wrapped_device_f_contact, wrapped_device_f_contact + f_contact_d.size());
  fApplied = DeviceValueArrayView(wrapped_device_fApplied, wrapped_device_fApplied + fApplied_d.size());
  fElastic = DeviceValueArrayView(wrapped_device_fElastic, wrapped_device_fElastic + fElastic_d.size());
  tmp = DeviceValueArrayView(wrapped_device_tmp, wrapped_device_tmp + tmp_d.size());
  r = DeviceValueArrayView(wrapped_device_r, wrapped_device_r + r_d.size());
  b = DeviceValueArrayView(wrapped_device_b, wrapped_device_b + b_d.size());
  k = DeviceValueArrayView(wrapped_device_k, wrapped_device_k + k_d.size());
  gamma = DeviceValueArrayView(wrapped_device_gamma, wrapped_device_gamma + gamma_d.size());

  // create mass matrix using cusp library (shouldn't change)
  thrust::device_ptr<int> wrapped_device_I(CASTI1(massI_d));
  DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + massI_d.size());

  thrust::device_ptr<int> wrapped_device_J(CASTI1(massJ_d));
  DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + massJ_d.size());

  thrust::device_ptr<double> wrapped_device_V(CASTD1(mass_d));
  DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + mass_d.size());

  mass = DeviceView(a_d.size(), a_d.size(), mass_d.size(), row_indices, column_indices, values);
  // end create mass matrix

  return 0;
}

int System::initializeSystem() {

  for(int j=0; j<bodies.size(); j++) {
    Body* body = bodies[j];
    body->setIdentifier(j); // Indicates the number that the Body was added
    body->setIndex(p_h.size()); // Indicates the Body's location in the position array

    // Push Body's location to global library
    indices_h.push_back(p_h.size());

    // update p
    p_h.push_back(body->pos.x);
    p_h.push_back(body->pos.y);
    p_h.push_back(body->pos.z);

    // update v
    v_h.push_back(body->vel.x);
    v_h.push_back(body->vel.y);
    v_h.push_back(body->vel.z);

    // update a
    a_h.push_back(body->acc.x);
    a_h.push_back(body->acc.y);
    a_h.push_back(body->acc.z);

    // update external force vector (gravity)
    if(body->isFixed()) {
      f_h.push_back(0);
      f_h.push_back(0);
      f_h.push_back(0);
    }
    else {
      f_h.push_back(body->mass * this->gravity.x);
      f_h.push_back(body->mass * this->gravity.y);
      f_h.push_back(body->mass * this->gravity.z);
    }

    f_contact_h.push_back(0);
    f_contact_h.push_back(0);
    f_contact_h.push_back(0);

    fApplied_h.push_back(0);
    fApplied_h.push_back(0);
    fApplied_h.push_back(0);

    fElastic_h.push_back(0);
    fElastic_h.push_back(0);
    fElastic_h.push_back(0);

    tmp_h.push_back(0);
    tmp_h.push_back(0);
    tmp_h.push_back(0);

    r_h.push_back(0);
    r_h.push_back(0);
    r_h.push_back(0);

    r_h.push_back(0);
    r_h.push_back(0);
    r_h.push_back(0);

    k_h.push_back(0);
    k_h.push_back(0);
    k_h.push_back(0);

    // update the mass matrix
    for(int i = 0; i < body->numDOF; i++) {
      massI_h.push_back(i + body->numDOF * j);
      massJ_h.push_back(i + body->numDOF * j);
      if(body->isFixed()) {
        mass_h.push_back(0);
      }
      else {
        mass_h.push_back(1.0/body->mass);
      }
    }

    contactGeometry_h.push_back(body->contactGeometry);
    collisionGeometry_h.push_back(body->contactGeometry);
    collisionMap_h.push_back(make_int3(body->getIdentifier(),0,body->getCollisionFamily()));

    if(body->isFixed()) fixedBodies_h.push_back(j);
  }

  for(int j=0; j<beams.size(); j++) {
    beams[j]->addBeam(j); //TODO: Make a function like this for body (makes code cleaner)
  }

  initializeDevice();
  solver->setup();

  return 0;
}

int System::DoTimeStep() {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Perform collision detection
  collisionDetector->generateAxisAlignedBoundingBoxes();
  collisionDetector->detectPossibleCollisions_spatialSubdivision();
  collisionDetector->detectCollisions();

  buildAppliedImpulseVector();
  if(collisionDetector->numCollisions) {
    // Set up the QOCC
    buildContactJacobian();
    buildSchurVector();

    // Solve the QOCC
    solver->solve();

    // Perform time integration (contacts)
    cusp::multiply(DT,gamma,f_contact);
    cusp::blas::axpby(k,f_contact,tmp,1.0,1.0);
    cusp::multiply(mass,tmp,v);
    cusp::blas::scal(f_contact,1.0/h);
  }
  else {
    // Perform time integration (no contacts)
    cusp::multiply(mass,k,v);

    cusp::blas::fill(f_contact,0.0);
  }
//  v_h = v_d;
//  v_h[3*bodies.size()+0] = 0;
//  v_h[3*bodies.size()+1] = 0;
//  v_h[3*bodies.size()+2] = 0;
//  v_d = v_h;
  cusp::blas::axpy(v, p, h);

  time += h;
  timeIndex++;
  //p_h = p_d;

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float execTime;
  hipEventElapsedTime(&execTime, start, stop);
  elapsedTime = execTime;

  printf("Time: %f (Exec. Time: %f), Collisions: %d (%d possible)\n",time,elapsedTime,collisionDetector->numCollisions, (int)collisionDetector->numPossibleCollisions);

  size_t avail;
  size_t total;
  hipMemGetInfo( &avail, &total );
  size_t used = total - avail;
  totalGPUMemoryUsed = used/1000000.0;
  cout << "  Device memory used: " << totalGPUMemoryUsed << " MB (Avail: " << avail/1000000 << " MB)" << endl;

  return 0;
}

int System::applyForce(Body* body, double3 force) {
  int index = body->getIndex();
  //cout << index << endl;

  fApplied_h[index]+=force.x;
  fApplied_h[index+1]+=force.y;
  fApplied_h[index+2]+=force.z;

  return 0;
}

int System::clearAppliedForces() {
  Thrust_Fill(fApplied_d,0.0);
  fApplied_h = fApplied_d;

  return 0;
}

__global__ void constructContactJacobian(int* nonzerosPerContact_d, int3* collisionMap, double3* geometries, double3* collisionGeometry, int* DI, int* DJ, double* D, double* friction, double4* normalsAndPenetrations, uint* collisionIdentifierA, uint* collisionIdentifierB, int* indices, int numBodies, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  friction[index] = 0.25; // TODO: EDIT THIS TO BE MINIMUM OF FRICTION COEFFICIENTS

  int offsetA = (!index) ? 0 : nonzerosPerContact_d[index - 1];
  DI = &DI[offsetA];
  DJ = &DJ[offsetA];
  D = &D[offsetA];

  int bodyIdentifierA = collisionMap[collisionIdentifierA[index]].x;
  int bodyIdentifierB = collisionMap[collisionIdentifierB[index]].x;

  int endA = (bodyIdentifierA<numBodies) ? 3 : 12;
  int endB = (bodyIdentifierB<numBodies) ? 3 : 12;

  int indexA = indices[bodyIdentifierA];
  int indexB = indices[bodyIdentifierB];

  double xiA = static_cast<double>(collisionMap[collisionIdentifierA[index]].y)/(static_cast<double>(geometries[bodyIdentifierA].z-1));
  double lA = geometries[bodyIdentifierA].y;

  double xiB = static_cast<double>(collisionMap[collisionIdentifierB[index]].y)/(static_cast<double>(geometries[bodyIdentifierB].z-1));
  double lB = geometries[bodyIdentifierB].y;

  double4 nAndP;
  double3 n, u, v;
  nAndP = normalsAndPenetrations[index];
  n = make_double3(nAndP.x,nAndP.y,nAndP.z);

  if(n.z != 0) {
    u = normalize(make_double3(1,0,-n.x/n.z));
  }
  else if(n.x != 0) {
    u = normalize(make_double3(-n.z/n.x,0,1));
  }
  else {
    u = normalize(make_double3(1,-n.x/n.y,0));
  }
  v = normalize(cross(n,u));

  // Add n, i indices
  int i;
  int end = endA;
  int j = 0;
  for(i=0;i<end;i++) {
    DI[i] = 3*index+0;
    DJ[i] = indexA+j;
    j++;
  }
  end+=endB;
  j = 0;
  for(;i<end;i++) {
    DI[i] = 3*index+0;
    DJ[i] = indexB+j;
    j++;
  }

  // Add u, i indices
  end+=endA;
  j = 0;
  for(;i<end;i++) {
    DI[i] = 3*index+1;
    DJ[i] = indexA+j;
    j++;
  }
  end+=endB;
  j = 0;
  for(;i<end;i++) {
    DI[i] = 3*index+1;
    DJ[i] = indexB+j;
    j++;
  }

  // Add v, i indices
  end+=endA;
  j = 0;
  for(;i<end;i++) {
    DI[i] = 3*index+2;
    DJ[i] = indexA+j;
    j++;
  }
  end+=endB;
  j = 0;
  for(;i<end;i++) {
    DI[i] = 3*index+2;
    DJ[i] = indexB+j;
    j++;
  }

  // Add n, values
  int startIndex = 0;
  if(bodyIdentifierA<numBodies) {
    D[startIndex+0] = n.x;
    D[startIndex+1] = n.y;
    D[startIndex+2] = n.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = n.x*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+1 ] = n.y*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+2 ] = n.z*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+3 ] = lA*n.x*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+4 ] = lA*n.y*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+5 ] = lA*n.z*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+6 ] = n.x*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+7 ] = n.y*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+8 ] = n.z*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+9 ] = -lA*n.x*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+10] = -lA*n.y*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+11] = -lA*n.z*(-xiA*xiA*xiA+xiA*xiA);
    startIndex+=12;
  }
  if(bodyIdentifierB<numBodies) {
    D[startIndex+0] = -n.x;
    D[startIndex+1] = -n.y;
    D[startIndex+2] = -n.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = -n.x*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+1 ] = -n.y*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+2 ] = -n.z*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+3 ] = -lB*n.x*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+4 ] = -lB*n.y*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+5 ] = -lB*n.z*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+6 ] = -n.x*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+7 ] = -n.y*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+8 ] = -n.z*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+9 ] = lB*n.x*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+10] = lB*n.y*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+11] = lB*n.z*(-xiB*xiB*xiB+xiB*xiB);
    startIndex+=12;
  }

  // Add u, values
  if(bodyIdentifierA<numBodies) {
    D[startIndex+0] = u.x;
    D[startIndex+1] = u.y;
    D[startIndex+2] = u.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = u.x*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+1 ] = u.y*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+2 ] = u.z*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+3 ] = lA*u.x*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+4 ] = lA*u.y*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+5 ] = lA*u.z*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+6 ] = u.x*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+7 ] = u.y*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+8 ] = u.z*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+9 ] = -lA*u.x*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+10] = -lA*u.y*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+11] = -lA*u.z*(-xiA*xiA*xiA+xiA*xiA);
    startIndex+=12;
  }
  if(bodyIdentifierB<numBodies) {
    D[startIndex+0] = -u.x;
    D[startIndex+1] = -u.y;
    D[startIndex+2] = -u.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = -u.x*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+1 ] = -u.y*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+2 ] = -u.z*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+3 ] = -lB*u.x*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+4 ] = -lB*u.y*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+5 ] = -lB*u.z*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+6 ] = -u.x*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+7 ] = -u.y*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+8 ] = -u.z*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+9 ] = lB*u.x*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+10] = lB*u.y*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+11] = lB*u.z*(-xiB*xiB*xiB+xiB*xiB);
    startIndex+=12;
  }

  // Add v, values
  if(bodyIdentifierA<numBodies) {
    D[startIndex+0] = v.x;
    D[startIndex+1] = v.y;
    D[startIndex+2] = v.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = v.x*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+1 ] = v.y*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+2 ] = v.z*(2.0*xiA*xiA*xiA-3.0*xiA*xiA+1.0);
    D[startIndex+3 ] = lA*v.x*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+4 ] = lA*v.y*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+5 ] = lA*v.z*(xiA*xiA*xiA-2.0*xiA*xiA+xiA);
    D[startIndex+6 ] = v.x*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+7 ] = v.y*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+8 ] = v.z*(-2.0*xiA*xiA*xiA+3.0*xiA*xiA);
    D[startIndex+9 ] = -lA*v.x*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+10] = -lA*v.y*(-xiA*xiA*xiA+xiA*xiA);
    D[startIndex+11] = -lA*v.z*(-xiA*xiA*xiA+xiA*xiA);
    startIndex+=12;
  }
  if(bodyIdentifierB<numBodies) {
    D[startIndex+0] = -v.x;
    D[startIndex+1] = -v.y;
    D[startIndex+2] = -v.z;
    startIndex+=3;
  } else {
    D[startIndex+0 ] = -v.x*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+1 ] = -v.y*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+2 ] = -v.z*(2.0*xiB*xiB*xiB-3.0*xiB*xiB+1.0);
    D[startIndex+3 ] = -lB*v.x*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+4 ] = -lB*v.y*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+5 ] = -lB*v.z*(xiB*xiB*xiB-2.0*xiB*xiB+xiB);
    D[startIndex+6 ] = -v.x*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+7 ] = -v.y*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+8 ] = -v.z*(-2.0*xiB*xiB*xiB+3.0*xiB*xiB);
    D[startIndex+9 ] = lB*v.x*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+10] = lB*v.y*(-xiB*xiB*xiB+xiB*xiB);
    D[startIndex+11] = lB*v.z*(-xiB*xiB*xiB+xiB*xiB);
    startIndex+=12;
  }
}

__global__ void updateNonzerosPerContact(int* nonzerosPerContact, int3* collisionMap, uint* collisionIdentifierA, uint* collisionIdentifierB, int numBodies, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  int numNonzeros = 0;
  int bodyIdentifierA = collisionMap[collisionIdentifierA[index]].x;
  int bodyIdentifierB = collisionMap[collisionIdentifierB[index]].x;

  if(bodyIdentifierA<numBodies) {
    numNonzeros+=9;
  }
  else {
    numNonzeros+=36;
  }

  if(bodyIdentifierB<numBodies) {
    numNonzeros+=9;
  }
  else {
    numNonzeros+=36;
  }

  nonzerosPerContact[index] = numNonzeros;
}

int System::buildContactJacobian() {
  // update nonzeros per contact
  int totalNonzeros = 0;
  nonzerosPerContact_d.resize(collisionDetector->numCollisions);
  updateNonzerosPerContact<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTI1(nonzerosPerContact_d), CASTI3(collisionMap_d), CASTU1(collisionDetector->collisionIdentifierA_d), CASTU1(collisionDetector->collisionIdentifierB_d), bodies.size(), collisionDetector->numCollisions);
  Thrust_Inclusive_Scan_Sum(nonzerosPerContact_d, totalNonzeros);

  DI_d.resize(totalNonzeros);
  DJ_d.resize(totalNonzeros);
  D_d.resize(totalNonzeros);
  friction_d.resize(collisionDetector->numCollisions);

  constructContactJacobian<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTI1(nonzerosPerContact_d), CASTI3(collisionMap_d), CASTD3(contactGeometry_d), CASTD3(collisionGeometry_d), CASTI1(DI_d), CASTI1(DJ_d), CASTD1(D_d), CASTD1(friction_d), CASTD4(collisionDetector->normalsAndPenetrations_d), CASTU1(collisionDetector->collisionIdentifierA_d), CASTU1(collisionDetector->collisionIdentifierB_d), CASTI1(indices_d), bodies.size(), collisionDetector->numCollisions);

  // create contact jacobian using cusp library
  thrust::device_ptr<int> wrapped_device_I(CASTI1(DI_d));
  DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + DI_d.size());

  thrust::device_ptr<int> wrapped_device_J(CASTI1(DJ_d));
  DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + DJ_d.size());

  thrust::device_ptr<double> wrapped_device_V(CASTD1(D_d));
  DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + D_d.size());

  D = DeviceView(3*collisionDetector->numCollisions, 3*bodies.size()+12*beams.size(), D_d.size(), row_indices, column_indices, values);
  // end create contact jacobian

  buildContactJacobianTranspose();

  return 0;
}

int System::buildContactJacobianTranspose() {
  DTI_d = DJ_d;
  DTJ_d = DI_d;
  DT_d = D_d;

  // create contact jacobian using cusp library
  thrust::device_ptr<int> wrapped_device_I(CASTI1(DTI_d));
  DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + DI_d.size());

  thrust::device_ptr<int> wrapped_device_J(CASTI1(DTJ_d));
  DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + DJ_d.size());

  thrust::device_ptr<double> wrapped_device_V(CASTD1(DT_d));
  DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + D_d.size());

  DT = DeviceView(3*bodies.size()+12*beams.size(), 3*collisionDetector->numCollisions, DT_d.size(), row_indices, column_indices, values);
  // end create contact jacobian

  DT.sort_by_row(); // TODO: Do I need this?

  return 0;
}

__global__ void multiplyByMass(double* massInv, double* src, double* dst, uint numDOF) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numDOF);

  double mass = massInv[index];
  if(mass) mass = 1.0/mass;
  dst[index] = mass*src[index];
}

__global__ void multiplyByBeamMass(double3* geometries, double3* materials, double* src, double* dst, uint numBodies, uint numBeams) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numBeams);

  double3 geometry = geometries[numBodies+index];
  double A = PI*geometry.x*geometry.x;
  double l = geometry.y;
  double rho = materials[index].x;

  uint offset = 3*numBodies+12*index;
  dst[offset+0 ] = (13.0*A*rho*src[0+offset])/35.0 + (9.0*A*rho*src[6+offset])/70.0 + (11.0*A*l*rho*src[3+offset])/210.0 - (13.0*A*l*rho*src[9 +offset])/420.0;
  dst[offset+1 ] = (13.0*A*rho*src[1+offset])/35.0 + (9.0*A*rho*src[7+offset])/70.0 + (11.0*A*l*rho*src[4+offset])/210.0 - (13.0*A*l*rho*src[10+offset])/420.0;
  dst[offset+2 ] = (13.0*A*rho*src[2+offset])/35.0 + (9.0*A*rho*src[8+offset])/70.0 + (11.0*A*l*rho*src[5+offset])/210.0 - (13.0*A*l*rho*src[11+offset])/420.0;
  dst[offset+3 ] = (A*l*l*rho*src[3+offset])/105.0 - (A*l*l*rho*src[9 +offset])/140.0 + (11.0*A*l*rho*src[0+offset])/210.0 + (13.0*A*l*rho*src[6+offset])/420.0;
  dst[offset+4 ] = (A*l*l*rho*src[4+offset])/105.0 - (A*l*l*rho*src[10+offset])/140.0 + (11.0*A*l*rho*src[1+offset])/210.0 + (13.0*A*l*rho*src[7+offset])/420.0;
  dst[offset+5 ] = (A*l*l*rho*src[5+offset])/105.0 - (A*l*l*rho*src[11+offset])/140.0 + (11.0*A*l*rho*src[2+offset])/210.0 + (13.0*A*l*rho*src[8+offset])/420.0;
  dst[offset+6 ] = (9.0*A*rho*src[0+offset])/70.0 + (13.0*A*rho*src[6+offset])/35.0 + (13.0*A*l*rho*src[3+offset])/420.0 - (11.0*A*l*rho*src[9 +offset])/210.0;
  dst[offset+7 ] = (9.0*A*rho*src[1+offset])/70.0 + (13.0*A*rho*src[7+offset])/35.0 + (13.0*A*l*rho*src[4+offset])/420.0 - (11.0*A*l*rho*src[10+offset])/210.0;
  dst[offset+8 ] = (9.0*A*rho*src[2+offset])/70.0 + (13.0*A*rho*src[8+offset])/35.0 + (13.0*A*l*rho*src[5+offset])/420.0 - (11.0*A*l*rho*src[11+offset])/210.0;
  dst[offset+9 ] = (A*l*l*rho*src[9 +offset])/105.0 - (A*l*l*rho*src[3+offset])/140.0 - (13.0*A*l*rho*src[0+offset])/420.0 - (11.0*A*l*rho*src[6+offset])/210.0;
  dst[offset+10] = (A*l*l*rho*src[10+offset])/105.0 - (A*l*l*rho*src[4+offset])/140.0 - (13.0*A*l*rho*src[1+offset])/420.0 - (11.0*A*l*rho*src[7+offset])/210.0;
  dst[offset+11] = (A*l*l*rho*src[11+offset])/105.0 - (A*l*l*rho*src[5+offset])/140.0 - (13.0*A*l*rho*src[2+offset])/420.0 - (11.0*A*l*rho*src[8+offset])/210.0;
}

int System::buildAppliedImpulseVector() {
  // build k
  updateElasticForces();
  multiplyByMass<<<BLOCKS(3*bodies.size()),THREADS>>>(CASTD1(mass_d), CASTD1(v_d), CASTD1(k_d), 3*bodies.size());
  multiplyByBeamMass<<<BLOCKS(beams.size()),THREADS>>>(CASTD3(contactGeometry_d), CASTD3(materialsBeam_d), CASTD1(v_d), CASTD1(k_d), bodies.size(), beams.size());
  //cusp::blas::axpy(fElastic,fApplied,-1.0); //TODO: Come up with a fix for applied forces
  cusp::blas::axpbypcz(f,fElastic,k,k,h,-h,1.0);

  return 0;
}

__global__ void buildStabilization(double* b, double4* normalsAndPenetrations, double timeStep, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double penetration = normalsAndPenetrations[index].w;

  b[3*index] = penetration/timeStep;
  b[3*index+1] = 0;
  b[3*index+2] = 0;
}

int System::buildSchurVector() {
  // build r
  r_d.resize(3*collisionDetector->numCollisions);
  b_d.resize(3*collisionDetector->numCollisions);
  // TODO: There's got to be a better way to do this...
  //r.resize(3*collisionDetector->numCollisions);
  thrust::device_ptr<double> wrapped_device_r(CASTD1(r_d));
  r = DeviceValueArrayView(wrapped_device_r, wrapped_device_r + r_d.size());
  thrust::device_ptr<double> wrapped_device_b(CASTD1(b_d));
  b = DeviceValueArrayView(wrapped_device_b, wrapped_device_b + b_d.size());
  cusp::multiply(mass,k,tmp);
  cusp::multiply(D,tmp,r);

  buildStabilization<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTD1(b_d), CASTD4(collisionDetector->normalsAndPenetrations_d), h, collisionDetector->numCollisions);
  cusp::blas::axpy(b,r,1.0);

  return 0;
}

int System::buildSchurMatrix() {
  // build N
  cusp::multiply(mass,DT,MinvDT);
  cusp::multiply(D,MinvDT,N);

  return 0;
}

__global__ void getNormalComponent(double* src, double* dst, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  dst[index] = src[3*index];
}

__global__ void calculateConeViolation(double* gamma, double* friction, double* dst, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double gamma_t = sqrt(pow(gamma[3*index+1],2.0)+pow(gamma[3*index+2],2.0));
  double coneViolation = friction[index]*gamma[3*index] - gamma_t; // TODO: Keep the friction indexing in mind for bilaterals
  if(coneViolation>0) coneViolation = 0;
  dst[index] = coneViolation;
}

double4 System::getCCPViolation() {
  double4 violationCCP = make_double4(0,0,0,0);

  if(collisionDetector->numCollisions) {
    // Build normal impulse vector, gamma_n
    thrust::device_vector<double> gamma_n_d;
    gamma_n_d.resize(collisionDetector->numCollisions);
    thrust::device_ptr<double> wrapped_device_gamma_n(CASTD1(gamma_n_d));
    DeviceValueArrayView gamma_n = DeviceValueArrayView(wrapped_device_gamma_n, wrapped_device_gamma_n + gamma_n_d.size());
    getNormalComponent<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTD1(gamma_d), CASTD1(gamma_n_d), collisionDetector->numCollisions);
    violationCCP.x = Thrust_Min(gamma_n_d);
    if(violationCCP.x > 0) violationCCP.x = 0;

    // Build normal velocity vector, v_n
    thrust::device_vector<double> tmp_gamma_d;
    tmp_gamma_d.resize(3*collisionDetector->numCollisions);
    thrust::device_ptr<double> wrapped_device_tmp_gamma(CASTD1(tmp_gamma_d));
    DeviceValueArrayView tmp_gamma = DeviceValueArrayView(wrapped_device_tmp_gamma, wrapped_device_tmp_gamma + tmp_gamma_d.size());

    thrust::device_vector<double> v_n_d;
    v_n_d.resize(collisionDetector->numCollisions);
    thrust::device_ptr<double> wrapped_device_v_n(CASTD1(v_n_d));
    DeviceValueArrayView v_n = DeviceValueArrayView(wrapped_device_v_n, wrapped_device_v_n + v_n_d.size());
    cusp::multiply(D,v,tmp_gamma);
    cusp::blas::axpy(b,tmp_gamma,1.0);
    getNormalComponent<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTD1(tmp_gamma_d), CASTD1(v_n_d), collisionDetector->numCollisions);
    violationCCP.y = Thrust_Min(v_n_d);
    if(violationCCP.y > 0) violationCCP.y = 0;

    // Check complementarity condition
    violationCCP.z = cusp::blas::dot(gamma_n,v_n);

    // Check friction cone condition
    calculateConeViolation<<<BLOCKS(collisionDetector->numCollisions),THREADS>>>(CASTD1(gamma_d), CASTD1(friction_d), CASTD1(v_n_d), collisionDetector->numCollisions);
    violationCCP.w = cusp::blas::nrm2(v_n);
  }

  return violationCCP;
}

int System::exportSystem(string filename) {
  ofstream filestream;
  filestream.open(filename.c_str());

  p_h = p_d;
  v_h = v_d;
  filestream << "0, " << bodies.size() << ", 0, " << endl;
  for (int i = 0; i < bodies.size(); i++) {
    filestream
        << i << ", "
        << bodies[i]->isFixed() << ", "
        << p_h[3*i] << ", "
        << p_h[3*i+1] << ", "
        << p_h[3*i+2] << ", "
        << "1, "
        << "0, "
        << "0, "
        << "0, "
        << v_h[3*i] << ", "
        << v_h[3*i+1] << ", "
        << v_h[3*i+2] << ", ";

        if(contactGeometry_h[i].y == 0) {
          filestream
            << "0, "
            << contactGeometry_h[i].x << ", ";
        }
        else {
          filestream
            << "2, "
            << contactGeometry_h[i].x << ", "
            << contactGeometry_h[i].y << ", "
            << contactGeometry_h[i].z << ", ";
        }
        filestream
          << "\n";
  }
  filestream.close();

  return 0;
}

int System::importSystem(string filename) {
  double3 pos;
  double3 vel;
  double3 geometry = make_double3(0,0,0);
  int isFixed;
  string temp_data;
  int numBodies;
  double blah;
  int index;
  int shape;

  ifstream ifile(filename.c_str());
  getline(ifile,temp_data);
  for(int i=0; i<temp_data.size(); ++i){
    if(temp_data[i]==','){temp_data[i]=' ';}
  }
  stringstream ss1(temp_data);
  ss1>>blah>>numBodies>>blah;

  Body* bodyPtr;
  for(int i=0; i<numBodies; i++) {
    getline(ifile,temp_data);
    for(int i=0; i<temp_data.size(); ++i){
      if(temp_data[i]==','){temp_data[i]=' ';}
    }
    stringstream ss(temp_data);
    ss>>index>>isFixed>>pos.x>>pos.y>>pos.z>>blah>>blah>>blah>>blah>>vel.x>>vel.y>>vel.z>>shape;
    if(shape == 0) {
      ss>>geometry.x;
      geometry.y = 0;
      geometry.z = 0;
    } else {
      ss>>geometry.x>>geometry.y>>geometry.z;
    }

    bodyPtr = new Body(pos);
    bodyPtr->setBodyFixed(isFixed);
    bodyPtr->setGeometry(geometry);
    bodyPtr->setVelocity(vel);
    if(shape == 0) {
      bodyPtr->setMass(2600*4.0*3.14159*pow(geometry.x,3.0)/3.0);
    } else {
      bodyPtr->setMass(1.0);
    }
    add(bodyPtr);
    //cout << index << " " << isFixed << " " << pos.x << " " << pos.y << " " << pos.z << " " << "1 0 0 0 " << vel.x << " " << vel.y << " " << vel.z << " " << shape << " " << geometry.x << " " << geometry.y << " " << geometry.z << endl;
  }

  return 0;
}

int System::exportMatrices(string directory) {

  string filename = directory + "/D.mtx";
  cusp::io::write_matrix_market_file(D, filename);

  filename = directory + "/Minv.mtx";
  cusp::io::write_matrix_market_file(mass, filename);

  filename = directory + "/r.mtx";
  cusp::io::write_matrix_market_file(r, filename);

  filename = directory + "/b.mtx";
  cusp::io::write_matrix_market_file(b, filename);

  filename = directory + "/k.mtx";
  cusp::io::write_matrix_market_file(k, filename);

  return 0;
}
