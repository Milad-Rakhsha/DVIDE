#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "System.cuh"

System::System()
{
  gravity = make_double3(0,-9.81,0);

	// spike stuff
	partitions = 1;
	solverOptions.safeFactorization = true;
	solverOptions.trackReordering = true;
	solverOptions.maxNumIterations = 5000;
	preconditionerUpdateModulus = -1; // the preconditioner updates every ___ time steps
	preconditionerMaxKrylovIterations = -1; // the preconditioner updates if Krylov iterations are greater than ____ iterations
	// end spike stuff

	timeIndex = 0;
	time = 0;
	simTime = 0;
}

void System::setSolverType(int solverType)
{
	switch(solverType) {
	case 0:
		solverOptions.solverType = spike::BiCGStab;
		break;
	case 1:
		solverOptions.solverType = spike::BiCGStab1;
		break;
	case 2:
		solverOptions.solverType = spike::BiCGStab2;
		break;
	case 3:
		solverOptions.solverType = spike::MINRES;
		break;
	}
}

void System::setPrecondType(int useSpike)
{
	solverOptions.precondType = useSpike ? spike::Spike : spike::None;
}

void System::setTimeStep(double step_size, double precision)
{
	h = step_size;

	// Set tolerance for Newton iteration based on the precision in positions
	// and integration step-size.
	double safety = 1;////0.5;
	tol = safety * precision / (h * h);

	// Set the tolerances for Krylov
	solverOptions.relTol = std::min(0.01 * tol, 1e-6);
	solverOptions.absTol = 1e-10;
}

void System::printSolverParams()
{
	printf("Step size: %e\n", h);
	printf("Newton tolerance: %e\n", tol);
	printf("Krylov relTol: %e  abdTol: %e\n", solverOptions.relTol, solverOptions.absTol);
	printf("Max. Krylov iterations: %d\n", solverOptions.maxNumIterations);
	printf("----------------------------\n");
}

int System::add(Element* element) {
	//add the element
	element->setIdentifier(elements.size());
	this->elements.push_back(*element);

	// update p
	p_h.push_back(element->pos.x);
	p_h.push_back(element->pos.y);
	p_h.push_back(element->pos.z);

  // update v
  v_h.push_back(element->vel.x);
  v_h.push_back(element->vel.y);
  v_h.push_back(element->vel.z);

  // update a
  a_h.push_back(element->acc.x);
  a_h.push_back(element->acc.y);
  a_h.push_back(element->acc.z);

	// update external force vector (gravity)
	f_h.push_back(element->mass * this->gravity.x);
	f_h.push_back(element->mass * this->gravity.y);
	f_h.push_back(element->mass * this->gravity.z);

	for (int i = 0; i < element->numDOF; i++) {
	  massI_h.push_back(i + element->numDOF * (elements.size() - 1));
		massJ_h.push_back(i + element->numDOF * (elements.size() - 1));
		mass_h.push_back(element->mass);
	}

	return elements.size();
}

int System::initializeDevice() {
	p_d = p_h;
	v_d = v_h;
	a_d = a_h;
	f_d = f_h;

	massI_d = massI_h;
	massJ_d = massJ_h;
	mass_d = mass_h;

	thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
	thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
	thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
	thrust::device_ptr<double> wrapped_device_f(CASTD1(f_d));

	p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
	v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
	a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
	f = DeviceValueArrayView(wrapped_device_f, wrapped_device_f + f_d.size());

	// create mass matrix using cusp library (shouldn't change)
	thrust::device_ptr<int> wrapped_device_I(CASTI1(massI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + massI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(massJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + massJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(mass_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + mass_d.size());

	mass = DeviceView(a_d.size(), a_d.size(), mass_d.size(), row_indices, column_indices, values);
	// end create mass matrix

	return 0;
}

int System::initializeSystem() {

	initializeDevice();

	// create and setup the Spike::GPU solver
	m_spmv = new MySpmv(mass);
	mySolver = new SpikeSolver(partitions, solverOptions);
	mySolver->setup(mass);

	bool success = mySolver->solve(*m_spmv, f, a);

	return 0;
}

int System::DoTimeStep() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//cusp::multiply(mass, f, a);
	bool success = mySolver->solve(*m_spmv, f, a);
	cusp::blas::axpy(a, v, h);
	cusp::blas::axpy(v, p, h);

  time += h;
  timeIndex++;
  p_h = p_d;

  printf("Time: %f\n",time);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	return 0;
}
