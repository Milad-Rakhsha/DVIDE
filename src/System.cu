#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "System.cuh"

System::System()
{
  gravity = make_double3(0,-9.81,0);
  tol = 1e-8;
  h = 1e-3;
  timeIndex = 0;
  time = 0;

	// spike stuff
	partitions = 1;
	solverOptions.safeFactorization = true;
	solverOptions.trackReordering = true;
	solverOptions.maxNumIterations = 5000;
	preconditionerUpdateModulus = -1; // the preconditioner updates every ___ time steps
	preconditionerMaxKrylovIterations = -1; // the preconditioner updates if Krylov iterations are greater than ____ iterations
	//mySolver = new SpikeSolver(partitions, solverOptions);
	//m_spmv = new MySpmv(mass);
  stepKrylovIterations = 0;
  precUpdated = 0;
	// end spike stuff

  collisionDetector = new CollisionDetector(this);
}

void System::setSolverType(int solverType)
{
	switch(solverType) {
	case 0:
		solverOptions.solverType = spike::BiCGStab;
		break;
	case 1:
		solverOptions.solverType = spike::BiCGStab1;
		break;
	case 2:
		solverOptions.solverType = spike::BiCGStab2;
		break;
	case 3:
		solverOptions.solverType = spike::MINRES;
		break;
	}
}

void System::setPrecondType(int useSpike)
{
	solverOptions.precondType = useSpike ? spike::Spike : spike::None;
}

void System::setTimeStep(double step_size, double precision)
{
	h = step_size;

	// Set tolerance for Newton iteration based on the precision in positions
	// and integration step-size.
	double safety = 1;////0.5;
	tol = safety * precision / (h * h);

	// Set the tolerances for Krylov
	solverOptions.relTol = std::min(0.01 * tol, 1e-6);
	solverOptions.absTol = 1e-10;
}

void System::printSolverParams()
{
	printf("Step size: %e\n", h);
	printf("Newton tolerance: %e\n", tol);
	printf("Krylov relTol: %e  abdTol: %e\n", solverOptions.relTol, solverOptions.absTol);
	printf("Max. Krylov iterations: %d\n", solverOptions.maxNumIterations);
	printf("----------------------------\n");
}

int System::add(Body* body) {
  // TODO: make this function general for any Body
	//add the element
  body->setIndex(p_h.size()); // Indicates the Body's location in the position array
  indices_h.push_back(p_h.size()); // Push Body's location to global library
  body->setIdentifier(bodies.size()); // Indicates the number that the Body was added
	bodies.push_back(body);

	// update p
	p_h.push_back(body->pos.x);
	p_h.push_back(body->pos.y);
	p_h.push_back(body->pos.z);

  // update v
  v_h.push_back(body->vel.x);
  v_h.push_back(body->vel.y);
  v_h.push_back(body->vel.z);

  // update a
  a_h.push_back(body->acc.x);
  a_h.push_back(body->acc.y);
  a_h.push_back(body->acc.z);

	// update external force vector (gravity)
  if(body->isFixed()) {
    f_h.push_back(0);
    f_h.push_back(0);
    f_h.push_back(0);
  }
  else {
    f_h.push_back(body->mass * this->gravity.x);
    f_h.push_back(body->mass * this->gravity.y);
    f_h.push_back(body->mass * this->gravity.z);
  }

  f_contact_h.push_back(0);
  f_contact_h.push_back(0);
  f_contact_h.push_back(0);

	// update the mass matrix
	for (int i = 0; i < body->numDOF; i++) {
	  //if(!body->isFixed()) {
      massI_h.push_back(i + body->numDOF * (bodies.size() - 1));
      massJ_h.push_back(i + body->numDOF * (bodies.size() - 1));
      mass_h.push_back(1.0/body->mass);
	  //}
	}

	contactGeometry_h.push_back(body->contactGeometry);

	return bodies.size();
}

int System::initializeDevice() {
  indices_d = indices_h;
	p_d = p_h;
	v_d = v_h;
	a_d = a_h;
	f_d = f_h;
	f_contact_d = f_contact_h;

	massI_d = massI_h;
	massJ_d = massJ_h;
	mass_d = mass_h;

	contactGeometry_d = contactGeometry_h;

	thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
	thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
	thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
	thrust::device_ptr<double> wrapped_device_f(CASTD1(f_d));
	thrust::device_ptr<double> wrapped_device_f_contact(CASTD1(f_contact_d));

	p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
	v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
	a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
	f = DeviceValueArrayView(wrapped_device_f, wrapped_device_f + f_d.size());
	f_contact = DeviceValueArrayView(wrapped_device_f_contact, wrapped_device_f_contact + f_contact_d.size());

	// create mass matrix using cusp library (shouldn't change)
	thrust::device_ptr<int> wrapped_device_I(CASTI1(massI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + massI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(massJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + massJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(mass_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + mass_d.size());

	mass = DeviceView(a_d.size(), a_d.size(), mass_d.size(), row_indices, column_indices, values);
	// end create mass matrix

	return 0;
}

int System::initializeSystem() {

  // update the contact geometry
  for(int i=0; i<bodies.size(); i++) {
    contactGeometry_h[i] = bodies[i]->contactGeometry;
  }
	initializeDevice();
	//collisionDetector->generateAxisAlignedBoundingBoxes_host();

	//collisionDetector->generateAxisAlignedBoundingBoxes();
	//collisionDetector->detectPossibleCollisions_spatialSubdivision();

	// create and setup the Spike::GPU solver
	//m_spmv = new MySpmv(mass);
	//mySolver = new SpikeSolver(partitions, solverOptions);
	//mySolver->setup(mass);

	//bool success = mySolver->solve(*m_spmv, f, a);

	//collisionDetector->detectPossibleCollisions_nSquared();

	return 0;
}

int System::DoTimeStep() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//cout << "Generate AABBs!" << endl;
	collisionDetector->generateAxisAlignedBoundingBoxes();
	collisionDetector->detectPossibleCollisions_spatialSubdivision();
  collisionDetector->detectCollisions();

  //cout << "Apply contact forces!" << endl;
  applyContactForces();
  cusp::blas::axpy(f, f_contact, 1.0);

  //cout << "Fix bodies!" << endl;
  fixBodies();

  //cout << "Integrate!" << endl;
	cusp::multiply(mass, f_contact, a);
	//bool success = mySolver->solve(*m_spmv, f, a);
	cusp::blas::axpy(a, v, h);
	cusp::blas::axpy(v, p, h);

  time += h;
  timeIndex++;
  p_h = p_d;

  printf("Time: %f, Collisions: %d\n",time,collisionDetector->collisionPairs_h.size());
  //cin.get();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	return 0;
}

int System::applyContactForces() {
  // TODO: Perform in parallel
  Thrust_Fill(f_contact_h,0);

  for(int i=0; i<collisionDetector->collisionPairs_h.size(); i++) {
    uint2 pairs = collisionDetector->collisionPairs_h[i];
    double3 normal = collisionDetector->normals_h[i];
    double penetration = collisionDetector->penetrations_h[i];

    double sigmaA = (1.0-0.25)/2.0e7;
    double sigmaB = sigmaA;
    double rA = 0.4;
    double rB = 0.4;
    double3 contactForce = 4.0/(3.0*(sigmaA+sigmaB))*sqrt(rA*rB/(rA+rB))*pow(penetration,1.5)*normal;

    // Add damping
    v_h = v_d;
    double3 v = make_double3(v_h[indices_h[pairs.y]]-v_h[indices_h[pairs.x]],v_h[indices_h[pairs.y]+1]-v_h[indices_h[pairs.x]+1],v_h[indices_h[pairs.y]+2]-v_h[indices_h[pairs.x]+2]);
    double b = 250;
    double3 damping;
    damping.x = b * normal.x * normal.x * v.x + b * normal.x * normal.y * v.y + b * normal.x * normal.z * v.z;
    damping.y = b * normal.x * normal.y * v.x + b * normal.y * normal.y * v.y + b * normal.y * normal.z * v.z;
    damping.z = b * normal.x * normal.z * v.x + b * normal.y * normal.z * v.y + b * normal.z * normal.z * v.z;
    contactForce -= damping;

    f_contact_h[indices_h[pairs.x]]   -= contactForce.x;
    f_contact_h[indices_h[pairs.x]+1] -= contactForce.y;
    f_contact_h[indices_h[pairs.x]+2] -= contactForce.z;

    f_contact_h[indices_h[pairs.y]]   += contactForce.x;
    f_contact_h[indices_h[pairs.y]+1] += contactForce.y;
    f_contact_h[indices_h[pairs.y]+2] += contactForce.z;

  }
  f_contact_d = f_contact_h;

  return 0;
}

int System::fixBodies() {
  f_contact_h = f_contact_d;
  for(int i=0; i<bodies.size(); i++) {
    if(bodies[i]->isFixed()) {
      f_contact_h[indices_h[i]]   = 0;
      f_contact_h[indices_h[i]+1] = 0;
      f_contact_h[indices_h[i]+2] = 0;
    }
  }
  f_contact_d = f_contact_h;

  return 0;
}


