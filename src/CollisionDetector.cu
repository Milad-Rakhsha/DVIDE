#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "CollisionDetector.cuh"

void CollisionDetector::setBinsPerAxis(uint3 binsPerAxis) {
  this->binsPerAxis = binsPerAxis;
}

inline uint3 __device__ getHashMin(const double3 &A, const double3 & binSizeInverse) {
  uint3 temp;
  temp.x = A.x * binSizeInverse.x;
  temp.y = A.y * binSizeInverse.y;
  temp.z = A.z * binSizeInverse.z;

  return temp;
}

inline uint3 __device__ getHashMax(const double3 &A, const double3 & binSizeInverse) {
  uint3 temp;
  temp.x = A.x * binSizeInverse.x;
  temp.y = A.y * binSizeInverse.y;
  temp.z = A.z * binSizeInverse.z;

  return temp;
}

inline uint __device__ getHashIndex(const uint3 &A, const uint3 &binsPerAxis) {
  //return ((A.x * 73856093) ^ (A.y * 19349663) ^ (A.z * 83492791));
  return A.x+A.y*binsPerAxis.x+A.z*binsPerAxis.x*binsPerAxis.y;
}

__global__ void countAabbBinIntersections(double3* aabbData, uint* numBinsIntersected, double3 binSizeInverse, uint numAABB) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numAABB);

  uint3 gmin = getHashMin(aabbData[index], binSizeInverse);
  uint3 gmax = getHashMax(aabbData[index + numAABB], binSizeInverse);
  //uint3 check = getHashMin(aabbData[index + numAABB]-aabbData[index],binSizeInverse);
  //check += make_uint3(1,1,1);
  //printf("AABB #[%d]: (%d-%d+1)*(%d-%d+1)*(%d-%d+1) or (%d*%d*%d)\n",index,gmax.x,gmin.x,gmax.y,gmin.y,gmax.z,gmin.z,check.x,check.y,check.z);
  numBinsIntersected[index] = (gmax.x - gmin.x + 1) * (gmax.y - gmin.y + 1) * (gmax.z - gmin.z + 1);
  //numBinsIntersected[index] = check.x*check.y*check.z;
}

__global__ void storeAabbBinIntersections(double3* aabbData, uint* numBinsIntersected, uint * binIdentifier, uint * aabbIdentifier, double3 binSizeInverse, uint3 binsPerAxis, uint numAABB) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numAABB);

  uint count = 0, i, j, k;
  uint3 gmin = getHashMin(aabbData[index], binSizeInverse);
  uint3 gmax = getHashMax(aabbData[index + numAABB], binSizeInverse);
  uint mInd = (index == 0) ? 0 : numBinsIntersected[index - 1];

  for (i = gmin.x; i <= gmax.x; i++) {
    for (j = gmin.y; j <= gmax.y; j++) {
      for (k = gmin.z; k <= gmax.z; k++) {
        binIdentifier[mInd + count] = getHashIndex(make_uint3(i, j, k),binsPerAxis);
        aabbIdentifier[mInd + count] = index;
        count++;
      }
    }
  }
}

__global__ void countAabbAabbIntersections(double3* aabbData, uint * binIdentifier, uint * aabbIdentifier, uint * binStartIndex, uint* numAabbCollisionsPerBin, uint lastActiveBin, uint numAABB) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, lastActiveBin);

  uint end = binStartIndex[index], count = 0, i = (!index) ? 0 : binStartIndex[index - 1];
  uint tempa, tempb;
  AABBstruct A, B;
  for (; i < end; i++) {
    tempa = aabbIdentifier[i];
    A.min = aabbData[tempa];
    A.max = aabbData[tempa + numAABB];
    for (int k = i + 1; k < end; k++) {
      tempb = aabbIdentifier[k];
      B.min = aabbData[tempb];
      B.max = aabbData[tempb + numAABB];
      bool inContact = (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
      if (inContact) count++;
    }
  }
  numAabbCollisionsPerBin[index] = count;
}

__global__ void storeAabbAabbIntersections(double3* aabbData, uint * binIdentifier, uint * aabbIdentifier, uint * binStartIndex, uint* Num_ContactD, long long* potentialCollisions, uint lastActiveBin, uint numAABB) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, lastActiveBin);

  uint end = binStartIndex[index], count = 0, i = (!index) ? 0 : binStartIndex[index - 1], Bin = binIdentifier[index];
  uint offset = (!index) ? 0 : Num_ContactD[index - 1];
  if (end - i == 1) {
    return;
  }
  uint tempa, tempb;
  AABBstruct A, B;
  for (; i < end; i++) {
    ;
    tempa = aabbIdentifier[i];
    A.min = aabbData[tempa];
    A.max = aabbData[tempa + numAABB];
    for (int k = i + 1; k < end; k++) {
      tempb = aabbIdentifier[k];
      B.min = aabbData[tempb];
      B.max = aabbData[tempb + numAABB];
      bool inContact = (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
      if (inContact) {
        int a = tempa;
        int b = tempb;
        if (b < a) {
          int t = a;
          a = b;
          b = t;
        }
        potentialCollisions[offset + count] = ((long long) a << 32 | (long long) b); //the two indices of the objects that make up the contact
        count++;
      }
    }
  }
}

CollisionDetector::CollisionDetector(System* sys)
{
  system = sys;
  numAABB = 0;
  binsPerAxis = make_uint3(20,20,20);
  numPossibleCollisions = 0;
  totalBinIntersections = 0;
  lastActiveBin = 0;
  possibleCollisionPairs_d.clear();
  collisionPairs_d.clear();
  numCollisions = 0;

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(countAabbBinIntersections), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(storeAabbBinIntersections), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(countAabbAabbIntersections), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(storeAabbAabbIntersections), hipFuncCachePreferL1);
}

int CollisionDetector::detectPossibleCollisions_nSquared()
{
  // Perform n-squared collision detection, only needs to be called once!
  for(int i=0; i<system->bodies.size(); i++) {
    for(int j=i+1; j<system->bodies.size(); j++) {
      possibleCollisionPairs_h.push_back(make_uint2(i,j));
    }
  }
  possibleCollisionPairs_d = possibleCollisionPairs_h;

  return 0;
}

__global__ void generateAabbData(double3* aabbData, int* indices, double* position, double3* geometries, uint numAABB) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numAABB);

  double3 pos = make_double3(position[indices[index]],position[indices[index]+1],position[indices[index]+2]);
  double3 geometry = geometries[index];
  if(geometry.y == 0) {
    // sphere case
    geometry = make_double3(geometry.x,geometry.x,geometry.x);
  }
  aabbData[index] = pos-geometry;
  aabbData[index + numAABB] = pos+geometry;
}

int CollisionDetector::generateAxisAlignedBoundingBoxes()
{
  aabbData_d.resize(2*system->bodies.size());
  generateAabbData<<<BLOCKS(system->bodies.size()),THREADS>>>(CASTD3(aabbData_d), CASTI1(system->indices_d), CASTD1(system->p_d), CASTD3(system->contactGeometry_d), system->bodies.size());

  return 0;
}

__global__ void convertLongsToInts(long long* potentialCollisions, uint2 * possibleCollisionPairs, uint numPossibleCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numPossibleCollisions);

  possibleCollisionPairs[index].x = int(potentialCollisions[index] >> 32);
  possibleCollisionPairs[index].y = int(potentialCollisions[index] & 0xffffffff);
}

int CollisionDetector::detectPossibleCollisions_spatialSubdivision()
{
  // Step 1: Initialize
  numAABB = aabbData_d.size()*0.5;
  possibleCollisionPairs_d.clear();
  // End Step 1

  // Step 2: Determine the bounds on the total space and subdivide based on the bins per axis
  double3 first = aabbData_d[0];//make_double3(0,0,0);
  AABB init = AABB(first, first); // create a zero volume AABB
  AABB_transformation unary_op;
  AABB_reduction binary_op;
  AABB result = thrust::transform_reduce(aabbData_d.begin(), aabbData_d.end(), unary_op, init, binary_op);
  minBoundingPoint = result.first-make_double3(0.01,0.01,0.01);
  maxBoundingPoint = result.second+make_double3(0.01,0.01,0.01);
  globalOrigin = minBoundingPoint;

  binSizeInverse.x = ((double)binsPerAxis.x)/fabs(maxBoundingPoint.x - minBoundingPoint.x);
  binSizeInverse.y = ((double)binsPerAxis.y)/fabs(maxBoundingPoint.y - minBoundingPoint.y);
  binSizeInverse.z = ((double)binsPerAxis.z)/fabs(maxBoundingPoint.z - minBoundingPoint.z);

  thrust::transform(aabbData_d.begin(), aabbData_d.end(), thrust::constant_iterator<double3>(globalOrigin), aabbData_d.begin(), thrust::minus<double3>());
  // End Step 2

  // Step 3: Count the number of AABB's that lie in each bin, allocate space for each AABB
  numBinsIntersected_d.resize(numAABB);

  // need to figure out how many bins each AABB intersects
  countAabbBinIntersections<<<BLOCKS(numAABB),THREADS>>>(CASTD3(aabbData_d), CASTU1(numBinsIntersected_d), binSizeInverse, numAABB);

  // need to use an inclusive scan to figure out where each thread should start entering the bin that each AABB is in (also counts total bin intersections)
  Thrust_Inclusive_Scan_Sum(numBinsIntersected_d, totalBinIntersections);

  binIdentifier_d.resize(totalBinIntersections);
  aabbIdentifier_d.resize(totalBinIntersections);
  binStartIndex_d.resize(totalBinIntersections);
  // End Step 3

  // Step 4: Indicate what bin each AABB belongs to, then sort based on bin number
  storeAabbBinIntersections<<<BLOCKS(numAABB),THREADS>>>(CASTD3(aabbData_d), CASTU1(numBinsIntersected_d), CASTU1(binIdentifier_d), CASTU1(aabbIdentifier_d), binSizeInverse, binsPerAxis, numAABB);

  // After figuring out which bin each AABB belongs to, sort the AABB's based on bin number
  Thrust_Sort_By_Key(binIdentifier_d, aabbIdentifier_d);

  // Next, count the number of AABB's that each bin has (this destroys the information in binIdentifier and puts it into aabbIdentifier)
  Thrust_Reduce_By_KeyA(lastActiveBin, binIdentifier_d, binStartIndex_d);

  binStartIndex_d.resize(lastActiveBin);

  // reduce the # of AABB's per bin to create a library so a thread knows where each bin starts and ends
  Thrust_Inclusive_Scan(binStartIndex_d);

  numAabbCollisionsPerBin_d.resize(lastActiveBin);
  // End Step 4

  // Step 5: Count the number of AABB collisions
  // At this point, binIdentifier has the bin number for each thread, binStartIndex tells the thread where to start and stop, and aabbIdentifier has the AABB that is in the bin
  countAabbAabbIntersections<<<BLOCKS(lastActiveBin),THREADS>>>(CASTD3(aabbData_d), CASTU1(binIdentifier_d), CASTU1(aabbIdentifier_d), CASTU1(binStartIndex_d), CASTU1(numAabbCollisionsPerBin_d), lastActiveBin, numAABB);

  Thrust_Inclusive_Scan_Sum(numAabbCollisionsPerBin_d, numPossibleCollisions);
  potentialCollisions_d.resize(numPossibleCollisions);
  // End Step 5

  // Step 6: Store the possible AABB collision pairs
  storeAabbAabbIntersections<<<BLOCKS(lastActiveBin),THREADS>>>(CASTD3(aabbData_d), CASTU1(binIdentifier_d), CASTU1(aabbIdentifier_d), CASTU1(binStartIndex_d), CASTU1(numAabbCollisionsPerBin_d), CASTLL(potentialCollisions_d), lastActiveBin, numAABB);
  thrust::sort(potentialCollisions_d.begin(), potentialCollisions_d.end());
  numPossibleCollisions = thrust::unique(potentialCollisions_d.begin(), potentialCollisions_d.end()) - potentialCollisions_d.begin();
  // End Step 6

  // Step 7: Convert long long potentialCollisions_d to int2 possibleCollisionPairs_d
  possibleCollisionPairs_d.resize(numPossibleCollisions);
  convertLongsToInts<<<BLOCKS(numPossibleCollisions),THREADS>>>(CASTLL(potentialCollisions_d), CASTU2(possibleCollisionPairs_d), numPossibleCollisions);
  // End Step 7

  return 0;
}

__global__ void countActualCollisions(uint* numCollisionsPerPair, uint2* possibleCollisionPairs, double* p, int* indices, double3* geometries, uint numPossibleCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numPossibleCollisions);

  int numCollisions = 0;

  int bodyA = possibleCollisionPairs[index].x;
  int bodyB = possibleCollisionPairs[index].y;

  double3 posA = make_double3(p[indices[bodyA]],p[indices[bodyA]+1],p[indices[bodyA]+2]);
  double3 posB = make_double3(p[indices[bodyB]],p[indices[bodyB]+1],p[indices[bodyB]+2]);

  double3 geometryA = geometries[bodyA];
  double3 geometryB = geometries[bodyB];

  if(geometryA.y == 0 && geometryB.y == 0) {
    // sphere-sphere case
    double penetration = (geometryA.x+geometryB.x) - length(posB-posA);
    if(penetration>0) {
      numCollisions++;
    }
  }
  else if(geometryA.y != 0 && geometryB.y == 0) {
    // box-sphere case
    numCollisions++;
  }
  else if(geometryA.y == 0 && geometryB.y != 0) {
    // sphere-box case
    numCollisions++;
  }
  else {
    // miscellaneous
  }
  numCollisionsPerPair[index] = numCollisions;
}

__global__ void storeActualCollisions(uint* numCollisionsPerPair, uint2* possibleCollisionPairs, double* p, int* indices, double3* geometries, double4* normalsAndPenetrations, uint* bodyIdentifiers, uint numPossibleCollisions, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numPossibleCollisions);

  uint startIndex = (index == 0) ? 0 : numCollisionsPerPair[index - 1];
  uint endIndex = numCollisionsPerPair[index];

  printf("Thread %d checks possible collisions pairs %d-%d\n",index,startIndex,endIndex);

  int count = 0;
  for (int i = startIndex; i < endIndex; i++) {
    int bodyA = possibleCollisionPairs[index+count].x;
    int bodyB = possibleCollisionPairs[index+count].y;

    double3 posA = make_double3(p[indices[bodyA]],p[indices[bodyA]+1],p[indices[bodyA]+2]);
    double3 posB = make_double3(p[indices[bodyB]],p[indices[bodyB]+1],p[indices[bodyB]+2]);

    double3 geometryA = geometries[bodyA];
    double3 geometryB = geometries[bodyB];

    printf("Thread %d: Body %d (%f, %f, %f) and %d (%f, %f, %f)\n  (%f, %f, %f)\n  (%f, %f, %f)\n",index,bodyA,posA.x,posA.y,posA.z,bodyB,posB.x,posB.y,posB.z,geometryA.x,geometryA.y,geometryA.z,geometryB.x,geometryB.y,geometryB.z);

    double3 normal;
    normal.x = 1;
    normal.y = 0;
    normal.z = 0;
    double penetration = 0;

    if(geometryA.y == 0 && geometryB.y == 0) {
      // sphere-sphere case
      penetration = (geometryA.x+geometryB.x) - length(posB-posA);
      normal = normalize(posB-posA); // from A to B!

      printf("Thread %d: Body %d and %d (%f, %f, %f) %f\n",index,bodyA,bodyB,normal.x,normal.y,normal.z,penetration);

    }

    else if(geometryA.y != 0 && geometryB.y == 0) {
      // box-sphere case
      // check x-face
      if((posB.y>=(posA.y-geometryA.y) && posB.y<=(posA.y+geometryA.y)) && (posB.z>=(posA.z-geometryA.z) && posB.z<=(posA.z+geometryA.z)))
      {
        normal = make_double3(posB.x-posA.x,0,0);
        penetration = (geometryB.x + geometryA.x) - fabs(posB.x-posA.x);
      }

      // check y
      else if((posB.x>=(posA.x-geometryA.x) && posB.x<=(posA.x+geometryA.x)) && (posB.z>=(posA.z-geometryA.z) && posB.z<=(posA.z+geometryA.z)))
      {
        normal = make_double3(0,posB.y-posA.y,0);
        penetration = (geometryB.x + geometryA.y) - fabs(posB.y-posA.y);
      }

      // check z
      else if((posB.x>=(posA.x-geometryA.x) && posB.x<=(posA.x+geometryA.x)) && (posB.y>=(posA.y-geometryA.y) && posB.y<=(posA.y+geometryA.y)))
      {
        normal = make_double3(0,0,posB.z-posA.z);
        penetration = (geometryB.x + geometryA.z) - fabs(posB.z-posA.z);
      }
    }

    else if(geometryA.y == 0 && geometryB.y != 0) {
      // sphere-box case
      // check x-face
      if((posA.y>=(posB.y-geometryB.y) && posA.y<=(posB.y+geometryB.y)) && (posA.z>=(posB.z-geometryB.z) && posA.z<=(posB.z+geometryB.z)))
      {
        normal = make_double3(posB.x-posA.x,0,0);
        penetration = (geometryB.x + geometryA.x) - fabs(posB.x-posA.x);
      }

      // check y
      else if((posA.x>=(posB.x-geometryB.x) && posA.x<=(posB.x+geometryB.x)) && (posA.z>=(posB.z-geometryB.z) && posA.z<=(posB.z+geometryB.z)))
      {
        normal = make_double3(0,posB.y-posA.y,0);
        penetration = (geometryB.y + geometryA.x) - fabs(posB.y-posA.y);
      }

      // check z
      else if((posA.x>=(posB.x-geometryB.x) && posA.x<=(posB.x+geometryB.x)) && (posA.y>=(posB.y-geometryB.y) && posA.y<=(posB.y+geometryB.y)))
      {
        normal = make_double3(0,0,posB.z-posA.z);
        penetration = (geometryB.z + geometryA.x) - fabs(posB.z-posA.z);
      }
    }

    bodyIdentifiers[i] = bodyA;
    normalsAndPenetrations[i] = make_double4(-normal.x,-normal.y,-normal.z,penetration); // from B to A!

    bodyIdentifiers[i+numCollisions] = bodyB;
    normalsAndPenetrations[i+numCollisions] = make_double4(normal.x,normal.y,normal.z,penetration); // from A to B!

    count++;
  }
}

int CollisionDetector::detectCollisions()
{
  // Step 1: Detect how many collisions actually occur between each pair
  numCollisionsPerPair_d.resize(numPossibleCollisions);
  countActualCollisions<<<BLOCKS(numPossibleCollisions),THREADS>>>(CASTU1(numCollisionsPerPair_d), CASTU2(possibleCollisionPairs_d), CASTD1(system->p_d), CASTI1(system->indices_d), CASTD3(system->contactGeometry_d), numPossibleCollisions);
  // End Step 1

  possibleCollisionPairs_h = possibleCollisionPairs_d;
  for(int i=0;i<numPossibleCollisions;i++) {
    printf("Actual Collisions between %d and %d: %d\n",possibleCollisionPairs_h[i].x,possibleCollisionPairs_h[i].y,(int)numCollisionsPerPair_d[i]);
  }
  printf("\n");

  // Step 2: Figure out where each thread needs to start and end for each collision
  Thrust_Inclusive_Scan_Sum(numCollisionsPerPair_d, numCollisions);
  normalsAndPenetrations_d.resize(2*numCollisions);
  bodyIdentifier_d.resize(2*numCollisions);
  // End Step 2

  possibleCollisionPairs_h = possibleCollisionPairs_d;
  for(int i=0;i<numPossibleCollisions;i++) {
    printf("Actual Collisions between %d and %d: %d\n",possibleCollisionPairs_h[i].x,possibleCollisionPairs_h[i].y,(int)numCollisionsPerPair_d[i]);
  }
  printf("Number of actual collisions: %d\n",numCollisions);

  // Step 3: Store the actual collisions
  storeActualCollisions<<<BLOCKS(numPossibleCollisions),THREADS>>>(CASTU1(numCollisionsPerPair_d), CASTU2(possibleCollisionPairs_d), CASTD1(system->p_d), CASTI1(system->indices_d), CASTD3(system->contactGeometry_d), CASTD4(normalsAndPenetrations_d), CASTU1(bodyIdentifier_d), numPossibleCollisions, numCollisions);
  // End Step 3

  thrust::host_vector<double4> normalsAndPenetrations_h = normalsAndPenetrations_d;
  for(int i=0;i<2*numCollisions;i++) {
    printf("Body %d: Normal: (%f, %f, %f), Penetration: %f\n",(int)bodyIdentifier_d[i],normalsAndPenetrations_h[i].x,normalsAndPenetrations_h[i].y,normalsAndPenetrations_h[i].z,normalsAndPenetrations_h[i].w);
  }
  printf("\n");

  // Step 4: Sort the collisions by body identifier
  Thrust_Sort_By_Key(bodyIdentifier_d, normalsAndPenetrations_d);
  // End Step 4

  normalsAndPenetrations_h = normalsAndPenetrations_d;
  for(int i=0;i<2*numCollisions;i++) {
    printf("Body %d: Normal: (%f, %f, %f), Penetration: %f\n",(int)bodyIdentifier_d[i],normalsAndPenetrations_h[i].x,normalsAndPenetrations_h[i].y,normalsAndPenetrations_h[i].z,normalsAndPenetrations_h[i].w);
  }
  printf("\n");

  // Step 5: Count the number of collisions that each body has and place into collisionStartIndex_d
  collisionStartIndex_d.resize(2*numCollisions);
  Thrust_Reduce_By_KeyA(lastActiveCollision, bodyIdentifier_d, collisionStartIndex_d);
  collisionStartIndex_d.resize(lastActiveCollision);
  bodyIdentifier_d.resize(lastActiveCollision);
  // End Step 5

  for(int i=0;i<2*numCollisions;i++) {
    printf("bodyIdentifier_d[%d] -> collisionStartIndex_d[%d]\n",(int)bodyIdentifier_d[i],(int)collisionStartIndex_d[i]);
  }
  printf("Last Active Collisions: %d\n\n",lastActiveCollision);

  // Step 6: Figure out where each thread needs to start and end for each collision
  Thrust_Inclusive_Scan(collisionStartIndex_d);
  // End Step 6

  for(int i=0;i<lastActiveCollision;i++) {
    printf("bodyIdentifier_d[%d] -> collisionStartIndex_d[%d]\n",(int)bodyIdentifier_d[i],(int)collisionStartIndex_d[i]);
  }
  cin.get();

  return 0;
}
/*
int CollisionDetector::detectCollisions_host()
{
  //TODO: Perform in parallel
  possibleCollisionPairs_h = possibleCollisionPairs_d; // need to do this in case we use spatial subdivision
  collisionPairs_h.clear();
  normals_h.clear();
  penetrations_h.clear();

  for(int i=0; i<numPossibleCollisions; i++) {
    int bodyA = possibleCollisionPairs_h[i].x;
    int bodyB = possibleCollisionPairs_h[i].y;

    // Both spheres
    if(system->contactGeometry_h[bodyA].y == 0 && system->contactGeometry_h[bodyB].y == 0) {
      double3 posA = make_double3(system->p_h[system->indices_h[bodyA]],system->p_h[system->indices_h[bodyA]+1],system->p_h[system->indices_h[bodyA]+2]);
      double3 posB = make_double3(system->p_h[system->indices_h[bodyB]],system->p_h[system->indices_h[bodyB]+1],system->p_h[system->indices_h[bodyB]+2]);
      double3 normal = normalize(posB-posA); // from A to B!
      double penetration = (system->contactGeometry_h[bodyA].x+system->contactGeometry_h[bodyB].x) - length(posB-posA);
      if(penetration>0) {
        collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
        normals_h.push_back(normal);
        penetrations_h.push_back(penetration);
      }
    }

    // A = Sphere, B = Box
    else if(system->contactGeometry_h[bodyA].y == 0 && system->contactGeometry_h[bodyB].y != 0) {
      double3 posA = make_double3(system->p_h[system->indices_h[bodyA]],system->p_h[system->indices_h[bodyA]+1],system->p_h[system->indices_h[bodyA]+2]);
      double3 posB = make_double3(system->p_h[system->indices_h[bodyB]],system->p_h[system->indices_h[bodyB]+1],system->p_h[system->indices_h[bodyB]+2]);

      // check x-face
      if((posA.y>=(posB.y-system->contactGeometry_h[bodyB].y) && posA.y<=(posB.y+system->contactGeometry_h[bodyB].y)) && (posA.z>=(posB.z-system->contactGeometry_h[bodyB].z) && posA.z<=(posB.z+system->contactGeometry_h[bodyB].z)))
      {
        double3 normal = make_double3(posB.x-posA.x,0,0);
        double penetration = (system->contactGeometry_h[bodyB].x + system->contactGeometry_h[bodyA].x) - fabs(posB.x-posA.x);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }

      // check y
      else if((posA.x>=(posB.x-system->contactGeometry_h[bodyB].x) && posA.x<=(posB.x+system->contactGeometry_h[bodyB].x)) && (posA.z>=(posB.z-system->contactGeometry_h[bodyB].z) && posA.z<=(posB.z+system->contactGeometry_h[bodyB].z)))
      {
        double3 normal = make_double3(0,posB.y-posA.y,0);
        double penetration = (system->contactGeometry_h[bodyB].y + system->contactGeometry_h[bodyA].x) - fabs(posB.y-posA.y);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }

      // check z
      else if((posA.x>=(posB.x-system->contactGeometry_h[bodyB].x) && posA.x<=(posB.x+system->contactGeometry_h[bodyB].x)) && (posA.y>=(posB.y-system->contactGeometry_h[bodyB].y) && posA.y<=(posB.y+system->contactGeometry_h[bodyB].y)))
      {
        double3 normal = make_double3(0,0,posB.z-posA.z);
        double penetration = (system->contactGeometry_h[bodyB].z + system->contactGeometry_h[bodyA].x) - fabs(posB.z-posA.z);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }

    }

    // A = Box, B = Sphere
    else if(system->contactGeometry_h[bodyA].y != 0 && system->contactGeometry_h[bodyB].y == 0) {
      double3 posA = make_double3(system->p_h[system->indices_h[bodyA]],system->p_h[system->indices_h[bodyA]+1],system->p_h[system->indices_h[bodyA]+2]);
      double3 posB = make_double3(system->p_h[system->indices_h[bodyB]],system->p_h[system->indices_h[bodyB]+1],system->p_h[system->indices_h[bodyB]+2]);

      // check x-face
      if((posB.y>=(posA.y-system->contactGeometry_h[bodyA].y) && posB.y<=(posA.y+system->contactGeometry_h[bodyA].y)) && (posB.z>=(posA.z-system->contactGeometry_h[bodyA].z) && posB.z<=(posA.z+system->contactGeometry_h[bodyA].z)))
      {
        double3 normal = make_double3(posB.x-posA.x,0,0);
        double penetration = (system->contactGeometry_h[bodyB].x + system->contactGeometry_h[bodyA].x) - fabs(posB.x-posA.x);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }

      // check y
      else if((posB.x>=(posA.x-system->contactGeometry_h[bodyA].x) && posB.x<=(posA.x+system->contactGeometry_h[bodyA].x)) && (posB.z>=(posA.z-system->contactGeometry_h[bodyA].z) && posB.z<=(posA.z+system->contactGeometry_h[bodyA].z)))
      {
        double3 normal = make_double3(0,posB.y-posA.y,0);
        double penetration = (system->contactGeometry_h[bodyB].x + system->contactGeometry_h[bodyA].y) - fabs(posB.y-posA.y);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }

      // check z
      else if((posB.x>=(posA.x-system->contactGeometry_h[bodyA].x) && posB.x<=(posA.x+system->contactGeometry_h[bodyA].x)) && (posB.y>=(posA.y-system->contactGeometry_h[bodyA].y) && posB.y<=(posA.y+system->contactGeometry_h[bodyA].y)))
      {
        double3 normal = make_double3(0,0,posB.z-posA.z);
        double penetration = (system->contactGeometry_h[bodyB].x + system->contactGeometry_h[bodyA].z) - fabs(posB.z-posA.z);
        if(penetration>0) {
          collisionPairs_h.push_back(make_uint2(bodyA,bodyB));
          normals_h.push_back(normalize(normal));
          penetrations_h.push_back(penetration);
        }
      }
    }
  }
  collisionPairs_d = collisionPairs_h;
  normals_d = normals_h;
  penetrations_d = penetrations_h;

  return 0;
}
*/
