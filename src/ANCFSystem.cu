#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "ANCFSystem.cuh"

#include <cusp/io/matrix_market.h>

//// linear operator y = A*x (for CUSP)
//class stencil: public cusp::linear_operator<double, cusp::device_memory> {
//public:
//        typedef cusp::linear_operator<double, cusp::device_memory> super;
//
//        int N;
//        DeviceView massMatrix;
//        DeviceView phiqMatrix;
//        DeviceValueArrayView temp;
//
//// constructor
//        stencil(int N, DeviceView lhs_mass, DeviceView lhs_phiq,
//                        DeviceValueArrayView tempVector) :
//                        super(N, N), N(N) {
//                massMatrix = lhs_mass;
//                phiqMatrix = lhs_phiq;
//                temp = tempVector;
//        }
//
//// linear operator y = A*x
//        template<typename VectorType1, typename VectorType2>
//        void operator()(const VectorType1& x, VectorType2& y) const {
//// obtain a raw pointer to device memory
//                cusp::multiply(massMatrix, x, temp);
//                cusp::multiply(phiqMatrix, x, y);
//                cusp::blas::axpy(temp, y, 1);
//        }
//};

ANCFSystem::ANCFSystem()
{
	// Set default solver parameters
	setAlpha_HHT(-0.1);
	setTimeStep(1e-3);
	maxNewtonIterations = 20;

	// spike stuff
	partitions = 1;
	solverOptions.safeFactorization = true;
	solverOptions.trackReordering = true;
	solverOptions.maxNumIterations = 5000;
	//mySpmv = new SpmvFunctor(lhs);
	// m_spmv = new MySpmv(lhs_mass, lhs_phiq, lhsVec);
	preconditionerUpdateModulus = -1; // the preconditioner updates every ___ time steps
	preconditionerMaxKrylovIterations = -1; // the preconditioner updates if Krylov iterations are greater than ____ iterations
	// end spike stuff

	this->timeIndex = 0;
	this->time = 0;
	timeToSimulate = 0;
	simTime = 0;
	fullJacobian = 1;

	wt3.push_back(5.0 / 9.0);
	wt3.push_back(8.0 / 9.0);
	wt3.push_back(5.0 / 9.0);

	pt3.push_back(-sqrt(3.0 / 5.0));
	pt3.push_back(0.0);
	pt3.push_back(sqrt(3.0 / 5.0));

	wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
	wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
	wt5.push_back(128. / 225.);
	wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
	wt5.push_back((322. - 13. * sqrt(70.)) / 900.);

	pt5.push_back(-(sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back(-(sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back(0.);
	pt5.push_back((sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back((sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);

	numCollisions = 0;
	numCollisionsSphere = 0;
	numContactPoints = 5;
	coefRestitution = .3;
	frictionCoef = .3;
	fileIndex = 0;

	// set up position files
	char filename1[100];
	char filename2[100];
	char filename3[100];
	sprintf(filename1, "position.dat");
	resultsFile1.open(filename1);
	sprintf(filename2, "energy.dat");
	resultsFile2.open(filename2);
	sprintf(filename3, "reactions.dat");
	resultsFile3.open(filename3);
}

void ANCFSystem::setSolverType(int solverType)
{
	switch(solverType) {
	case 0:
		solverOptions.solverType = spike::BiCGStab;
		break;
	case 1:
		solverOptions.solverType = spike::BiCGStab1;
		break;
	case 2:
		solverOptions.solverType = spike::BiCGStab2;
		break;
	case 3:
		solverOptions.solverType = spike::MINRES;
		break;
	}
}

void ANCFSystem::setPrecondType(int useSpike)
{
	solverOptions.precondType = useSpike ? spike::Spike : spike::None;
}

void ANCFSystem::setAlpha_HHT(double alpha) {
	// should be greater than -.3, usually set to -.1
	alphaHHT = alpha;
	betaHHT = (1 - alphaHHT) * (1 - alphaHHT) * .25;
	gammaHHT = 0.5 - alphaHHT;
}

void ANCFSystem::setTimeStep(double step_size,
                             double precision)
{
	h = step_size;

	// Set tolerance for Newton iteration based on the precision in positions
	// and integration step-size.
	double safety = 1;////0.5;
	tol = safety * precision / (h * h);

	// Set the tolerances for Krylov
	solverOptions.relTol = std::min(0.01 * tol, 1e-6);
	solverOptions.absTol = 1e-10;
}

void ANCFSystem::printSolverParams()
{
	printf("Step size: %e\n", h);
	printf("Newton tolerance: %e\n", tol);
	printf("Max. Newton iterations: %d\n", maxNewtonIterations);
	printf("Krylov relTol: %e  abdTol: %e\n", solverOptions.relTol, solverOptions.absTol);
	printf("Max. Krylov iterations: %d\n", solverOptions.maxNumIterations);
	printf("----------------------------\n");
}


int ANCFSystem::addParticle(Particle* particle) {
	//add the element
	particle->setParticleIndex(particles.size());
	this->particles.push_back(*particle);

	MaterialParticle material;
	material.E = particle->getElasticModulus();
	material.nu = particle->getNu();
	material.mass = particle->getMass();
	material.massInverse = 1.0 / particle->getMass();
	material.r = particle->getRadius();
	material.numContactPoints = 1;
	this->pMaterials_h.push_back(material);

	// update p
	float3 pos0 = particle->getInitialPosition();
	pParticle_h.push_back(pos0.x);
	pParticle_h.push_back(pos0.y);
	pParticle_h.push_back(pos0.z);

	// update v
	float3 vel0 = particle->getInitialVelocity();
	vParticle_h.push_back(vel0.x);
	vParticle_h.push_back(vel0.y);
	vParticle_h.push_back(vel0.z);

	for (int i = 0; i < 3; i++) {
		aParticle_h.push_back(0.0);
		fParticle_h.push_back(0.0);
	}

	return particles.size();
}

int ANCFSystem::addElement(Element* element) {
	//add the element
	element->setElementIndex(elements.size());
	this->elements.push_back(*element);

	Material material;
	material.E = element->getElasticModulus();
	material.l = element->getLength_l();
	material.nu = element->getNu();
	material.rho = element->getDensity();
	material.r = element->getRadius();
	material.numContactPoints = numContactPoints;
	this->materials.push_back(material);

	// update p
	Node node = element->getNode0();
	p_h.push_back(node.x);
	p_h.push_back(node.y);
	p_h.push_back(node.z);
	p_h.push_back(node.dx1);
	p_h.push_back(node.dy1);
	p_h.push_back(node.dz1);
	node = element->getNode1();
	p_h.push_back(node.x);
	p_h.push_back(node.y);
	p_h.push_back(node.z);
	p_h.push_back(node.dx1);
	p_h.push_back(node.dy1);
	p_h.push_back(node.dz1);

	for (int i = 0; i < 12; i++) {
		e_h.push_back(0.0);
		v_h.push_back(0.0);
		a_h.push_back(0.0);
		anew_h.push_back(0.0);
		fint_h.push_back(0.0);
		fcon_h.push_back(0.0);
		fapp_h.push_back(0.0);
		phiqlam_h.push_back(0.0);
		delta_h.push_back(0.0);
		strainDerivative_h.push_back(0.0);
	}
	strain_h.push_back(0.0);

	for (int i = 0; i < 4; i++) {
		Sx_h.push_back(0.0);
		Sxx_h.push_back(0.0);
	}

	//update other vectors (no initial velocity or acceleration)
	double r = element->getRadius();
	double a = element->getLength_l();
	double rho = element->getDensity();
	double A = PI * r * r;

	// update external force vector (gravity)
	fext_h.push_back(rho * A * a * GRAVITYx / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYy / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYz / 0.2e1);
	fext_h.push_back(rho * A * a * a * GRAVITYx / 0.12e2);
	fext_h.push_back(rho * A * a * a * GRAVITYy / 0.12e2);
	fext_h.push_back(rho * A * a * a * GRAVITYz / 0.12e2);
	fext_h.push_back(rho * A * a * GRAVITYx / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYy / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYz / 0.2e1);
	fext_h.push_back(-rho * A * a * a * GRAVITYx / 0.12e2);
	fext_h.push_back(-rho * A * a * a * GRAVITYy / 0.12e2);
	fext_h.push_back(-rho * A * a * a * GRAVITYz / 0.12e2);

	for (int i = 0; i < 12; i++) {
		for (int j = 0; j < 12; j++) {
			lhsI_h.push_back(i + 12 * (elements.size() - 1));
			lhsJ_h.push_back(j + 12 * (elements.size() - 1));
			lhs_h.push_back(0.0);
		}
	}

	return elements.size();
}

int ANCFSystem::addForce(Element* element, double xi, float3 force) {
	int index = element->getElementIndex();
	int l = element->getLength_l();

	//fapp_h = fapp_d;

	fapp_h[0 + 12 * index] += (1 - 3 * xi * xi + 2 * pow(xi, 3)) * force.x;
	fapp_h[1 + 12 * index] += (1 - 3 * xi * xi + 2 * pow(xi, 3)) * force.y;
	fapp_h[2 + 12 * index] += (1 - 3 * xi * xi + 2 * pow(xi, 3)) * force.z;
	fapp_h[3 + 12 * index] += l * (xi - 2 * xi * xi + pow(xi, 3)) * force.x;
	fapp_h[4 + 12 * index] += l * (xi - 2 * xi * xi + pow(xi, 3)) * force.y;
	fapp_h[5 + 12 * index] += l * (xi - 2 * xi * xi + pow(xi, 3)) * force.z;
	fapp_h[6 + 12 * index] += (3 * xi * xi - 2 * pow(xi, 3)) * force.x;
	fapp_h[7 + 12 * index] += (3 * xi * xi - 2 * pow(xi, 3)) * force.y;
	fapp_h[8 + 12 * index] += (3 * xi * xi - 2 * pow(xi, 3)) * force.z;
	fapp_h[9 + 12 * index] += l * (-xi * xi + pow(xi, 3)) * force.x;
	fapp_h[10 + 12 * index] += l * (-xi * xi + pow(xi, 3)) * force.y;
	fapp_h[11 + 12 * index] += l * (-xi * xi + pow(xi, 3)) * force.z;

	fapp_d = fapp_h;

	return 0;
}

int ANCFSystem::clearAppliedForces() {
	thrust::fill(fapp_d.begin(), fapp_d.end(), 0.0); //Clear internal forces
	return 0;
}

int ANCFSystem::updatePhiq() // used in Newton iteration, nice to keep it separate (but not memory efficient) - only needs to be done once (linear constraints)
{
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];

		phiqJ_h.push_back(i);
		phiqI_h.push_back(constraint.dofLoc.x);
		phiq_h.push_back(1.0);

		if (constraint.nodeNum2 != -1) {
			phiqJ_h.push_back(i);
			phiqI_h.push_back(constraint.dofLoc.y);
			phiq_h.push_back(-1.0);
		}
	}
	phiqI_d = phiqI_h;
	phiqJ_d = phiqJ_h;
	phiq_d = phiq_h;

	thrust::device_ptr<int> wrapped_device_I(CASTI1(phiqI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I,
			wrapped_device_I + phiqI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(phiqJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J,
			wrapped_device_J + phiqJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(phiq_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V,
			wrapped_device_V + phiq_d.size());

	phiq = DeviceView(12 * elements.size(), constraints.size(), phiq_d.size(),
			row_indices, column_indices, values);
	phiq.sort_by_row();

	return 0;
}

__global__ void calculateRHSlower(double* phi, double* p, double* phi0,
		double factor, int2* constraintPairs, int numConstraints) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < numConstraints) {
		int2 constraintPair = constraintPairs[i];
		if (constraintPair.y == -1) {
			phi[i] = factor * (p[constraintPair.x] - phi0[i]);
		} else {
			phi[i] = factor
					* (p[constraintPair.x] - p[constraintPair.y] - phi0[i]);
		}
		__syncthreads();
	}
}

int ANCFSystem::updatePhi() {
	calculateRHSlower<<<dimGridConstraint,dimBlockConstraint>>>(CASTD1(phi_d), CASTD1(pnew_d), CASTD1(phi0_d), 1.0/(betaHHT*h*h), CASTI2(constraintPairs_d), constraints.size());

	return 0;
}

__global__ void updateParticleDynamics_GPU(double h, double* a, double* v,
		double* p, double* f, MaterialParticle* materials, int numParticles) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < numParticles) {
		a = &a[3 * i];
		v = &v[3 * i];
		p = &p[3 * i];
		f = &f[3 * i];
		MaterialParticle material = materials[i];

		a[0] = material.massInverse * f[0] + GRAVITYx;
		a[1] = material.massInverse * f[1] + GRAVITYy;
		a[2] = material.massInverse * f[2] + GRAVITYz;

		for (int j = 0; j < 3; j++) {
			v[j] += h * a[j];
			p[j] += h * v[j];
		}
	}
}

int ANCFSystem::updateParticleDynamics() {
	updateParticleDynamics_GPU<<<dimGridParticles,dimBlockParticles>>>(h,CASTD1(aParticle_d), CASTD1(vParticle_d), CASTD1(pParticle_d), CASTD1(fParticle_d), CASTMP(pMaterials_d), particles.size());

	return 0;
}

int ANCFSystem::calculateInitialPhi() {
	for (int i = 0; i < constraints.size(); i++)
		phi0_h.push_back(0);
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];

		if (constraint.nodeNum2 == -1) {
			phi0_h[i] = p_h[constraint.dofLoc.x];
		} else {
			phi0_h[i] = p_h[constraint.dofLoc.x] - p_h[constraint.dofLoc.y];
		}
	}

	return 0;
}

int ANCFSystem::initializeDevice() {
	pMaterials_d = pMaterials_h;
	pParticle_d = pParticle_h;
	vParticle_d = vParticle_h;
	aParticle_d = aParticle_h;
	fParticle_d = fParticle_h;

	materials_d = materials;
	strainDerivative_d = strainDerivative_h;
	curvatureDerivative_d = strainDerivative_h;
	strain_d = strain_h;
	Sx_d = Sx_h;
	Sxx_d = Sxx_h;

	e_d = e_h;
	p_d = p_h;
	v_d = v_h;
	a_d = a_h;
	pnew_d = p_h;
	vnew_d = v_h;
	anew_d = anew_h;
	fext_d = fext_h;
	fint_d = fint_h;
	fapp_d = fapp_h;
	fcon_d = fcon_h;
	phi_d = phi_h;
	phi0_d = phi0_h;
	phiqlam_d = phiqlam_h;
	delta_d = delta_h;
	constraintPairs_d = constraintPairs_h;
	lhsVec_d = anew_h;

	lhsI_d = lhsI_h;
	lhsJ_d = lhsJ_h;
	lhs_d = lhs_h;

	constraintsI_d = constraintsI_h;
	constraintsJ_d = constraintsJ_h;
	constraints_d = constraints_h;

	thrust::device_ptr<double> wrapped_device_e(CASTD1(e_d));
	thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
	thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
	thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
	thrust::device_ptr<double> wrapped_device_pnew(CASTD1(pnew_d));
	thrust::device_ptr<double> wrapped_device_vnew(CASTD1(vnew_d));
	thrust::device_ptr<double> wrapped_device_anew(CASTD1(anew_d));
	thrust::device_ptr<double> wrapped_device_fext(CASTD1(fext_d));
	thrust::device_ptr<double> wrapped_device_fint(CASTD1(fint_d));
	thrust::device_ptr<double> wrapped_device_fapp(CASTD1(fapp_d));
	thrust::device_ptr<double> wrapped_device_fcon(CASTD1(fcon_d));
	thrust::device_ptr<double> wrapped_device_phi(CASTD1(phi_d));
	thrust::device_ptr<double> wrapped_device_phi0(CASTD1(phi0_d));
	thrust::device_ptr<double> wrapped_device_phiqlam(CASTD1(phiqlam_d));
	thrust::device_ptr<double> wrapped_device_delta(CASTD1(delta_d));
	thrust::device_ptr<double> wrapped_device_lhsVec(CASTD1(lhsVec_d));

	eAll = DeviceValueArrayView(wrapped_device_e,
			wrapped_device_e + e_d.size());
	eTop = DeviceValueArrayView(wrapped_device_e,
			wrapped_device_e + 12 * elements.size());
	eBottom = DeviceValueArrayView(wrapped_device_e + 12 * elements.size(),
			wrapped_device_e + e_d.size());
	p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
	v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
	a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
	pnew = DeviceValueArrayView(wrapped_device_pnew,
			wrapped_device_pnew + pnew_d.size());
	vnew = DeviceValueArrayView(wrapped_device_vnew,
			wrapped_device_vnew + vnew_d.size());
	anewAll = DeviceValueArrayView(wrapped_device_anew,
			wrapped_device_anew + anew_d.size());
	anew = DeviceValueArrayView(wrapped_device_anew,
			wrapped_device_anew + 12 * elements.size());
	lambda = DeviceValueArrayView(wrapped_device_anew + 12 * elements.size(),
			wrapped_device_anew + anew_d.size());
	fext = DeviceValueArrayView(wrapped_device_fext,
			wrapped_device_fext + fext_d.size());
	fint = DeviceValueArrayView(wrapped_device_fint,
			wrapped_device_fint + fint_d.size());
	fapp = DeviceValueArrayView(wrapped_device_fapp,
			wrapped_device_fapp + fapp_d.size());
	fcon = DeviceValueArrayView(wrapped_device_fcon,
			wrapped_device_fcon + fcon_d.size());
	phi = DeviceValueArrayView(wrapped_device_phi,
			wrapped_device_phi + phi_d.size());
	phi0 = DeviceValueArrayView(wrapped_device_phi0,
			wrapped_device_phi0 + phi0_d.size());
	phiqlam = DeviceValueArrayView(wrapped_device_phiqlam,
			wrapped_device_phiqlam + phiqlam_d.size());
	delta = DeviceValueArrayView(wrapped_device_delta,
			wrapped_device_delta + delta_d.size());
	lhsVec = DeviceValueArrayView(wrapped_device_lhsVec,
			wrapped_device_lhsVec + lhsVec_d.size());



	// create lhs matrix using cusp library (shouldn't change)
	thrust::device_ptr<int> wrapped_device_I(CASTI1(lhsI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I,
			wrapped_device_I + lhsI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(lhsJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J,
			wrapped_device_J + lhsJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(lhs_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V,
			wrapped_device_V + lhs_d.size());

	lhs = DeviceView(anew_d.size(), anew_d.size(), lhs_d.size(), row_indices,
			column_indices, values);
	// end create lhs matrix

	// create the view to the mass block of the lhs matrix
	DeviceIndexArrayView row_indices_mass = DeviceIndexArrayView(
			wrapped_device_I, wrapped_device_I + 12 * 12 * elements.size());
	DeviceIndexArrayView column_indices_mass = DeviceIndexArrayView(
			wrapped_device_J, wrapped_device_J + 12 * 12 * elements.size());
	DeviceValueArrayView values_mass = DeviceValueArrayView(wrapped_device_V,
			wrapped_device_V + 12 * 12 * elements.size());
	lhs_mass = DeviceView(anew_d.size(), anew_d.size(),
			12 * 12 * elements.size(), row_indices_mass, column_indices_mass,
			values_mass);
	// end create the view to the mass block of the lhs matrix

	// create the view to the mass block of the lhs matrix
	DeviceIndexArrayView row_indices_phiq = DeviceIndexArrayView(
			wrapped_device_I + 12 * 12 * elements.size(),
			wrapped_device_I + lhsI_d.size());
	DeviceIndexArrayView column_indices_phiq = DeviceIndexArrayView(
			wrapped_device_J + 12 * 12 * elements.size(),
			wrapped_device_J + lhsJ_d.size());
	DeviceValueArrayView values_phiq = DeviceValueArrayView(
			wrapped_device_V + 12 * 12 * elements.size(),
			wrapped_device_V + lhs_d.size());
	lhs_phiq = DeviceView(anew_d.size(), anew_d.size(),
			lhs_d.size() - 12 * 12 * elements.size(), row_indices_phiq,
			column_indices_phiq, values_phiq);
	lhs_phiq.sort_by_row(); // MUST BE SORTED FOR SPMV TO WORK CORRECTLY
	// end create the view to the mass block of the lhs matrix

	dimBlockConstraint.x = BLOCKDIMCONSTRAINT;
	dimGridConstraint.x = static_cast<int>(ceil(
			(static_cast<double>(constraints.size()))
					/ (static_cast<double>(BLOCKDIMCONSTRAINT))));

	dimBlockElement.x = BLOCKDIMELEMENT;
	dimGridElement.x = (int) ceil(
			((double) (elements.size())) / ((double) BLOCKDIMELEMENT));

	dimBlockParticles.x = BLOCKDIMELEMENT;
	dimGridParticles.x = (int) ceil(
			((double) (particles.size())) / ((double) BLOCKDIMELEMENT));

	dimBlockCollision.x = BLOCKDIMCOLLISION;
	dimGridCollision.x = (int) ceil(
			((double) (particles.size())) / ((double) BLOCKDIMCOLLISION));

	return 0;
}

int ANCFSystem::initializeSystem() {
	ANCFSystem::updatePhiq();
	ANCFSystem::calculateInitialPhi();

	for (int i = 0; i < constraints.size(); i++) {
		delta_h.push_back(0);
		e_h.push_back(0);
		anew_h.push_back(0);
		phi_h.push_back(0);
		constraintPairs_h.push_back(constraints[i].dofLoc);
	}

	// join phi_q to lhs
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];
		lhsI_h.push_back(i + 12 * elements.size());
		lhsJ_h.push_back(constraint.dofLoc.x);
		lhs_h.push_back(1.0);

		if (constraint.nodeNum2 != -1) {
			lhsI_h.push_back(i + 12 * elements.size());
			lhsJ_h.push_back(constraint.dofLoc.y);
			lhs_h.push_back(-1.0);
		}
	}

	// join phi_q' to lhs
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];
		lhsJ_h.push_back(i + 12 * elements.size());
		lhsI_h.push_back(constraint.dofLoc.x);
		lhs_h.push_back(1.0);

		if (constraint.nodeNum2 != -1) {
			lhsJ_h.push_back(i + 12 * elements.size());
			lhsI_h.push_back(constraint.dofLoc.y);
			lhs_h.push_back(-1.0);
		}
	}

	// Get constraints
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];
		constraintsI_h.push_back(i + 12 * elements.size());
		constraintsJ_h.push_back(constraint.dofLoc.x);
		constraints_h.push_back(1.0);

		if (constraint.nodeNum2 != -1) {
			constraintsI_h.push_back(i + 12 * elements.size());
			constraintsJ_h.push_back(constraint.dofLoc.y);
			constraints_h.push_back(-1.0);
		}
	}

	// join phi_q' to lhs
	for (int i = 0; i < constraints.size(); i++) {
		Constraint constraint = constraints[i];
		constraintsJ_h.push_back(i + 12 * elements.size());
		constraintsI_h.push_back(constraint.dofLoc.x);
		constraints_h.push_back(1.0);

		if (constraint.nodeNum2 != -1) {
			constraintsJ_h.push_back(i + 12 * elements.size());
			constraintsI_h.push_back(constraint.dofLoc.y);
			constraints_h.push_back(-1.0);
		}
	}

	initializeDevice();
	//ANCFSystem::initializeBoundingBoxes_CPU();
	//detector.updateBoundingBoxes(aabb_data_d);
	//detector.setBoundingBoxPointer(&aabb_data_d);
	//detector.detectPossibleCollisions();

	ANCFSystem::resetLeftHandSideMatrix();
	ANCFSystem::updateInternalForces();

	//cusp::blas::axpy(fint,eTop,-1);
	cusp::blas::axpby(fext, fint, eTop, 1, -1);

	// create and setup the Spike::GPU solver
	m_spmv = new MySpmv(lhs_mass, lhs_phiq, lhsVec);
	mySolver = new SpikeSolver(partitions, solverOptions);
	mySolver->setup(lhs);

//	char filename[100];
//	sprintf(filename, "./lhs.txt");
//	cusp::io::write_matrix_market_file(lhs, filename);
	cusp::blas::fill(anewAll, 0);
	bool success = mySolver->solve(*m_spmv, eAll, anewAll);
	spike::Stats stats = mySolver->getStats();
//	cout << "Success: " << success << " Iterations: "
//			<< stats.numIterations << " relResidualNorm: "
//			<< stats.relResidualNorm << endl;

	cout << endl
	     << "Linear problem size:  " << eAll.size() << endl
	     << "Number partitions:    " << stats.numPartitions << endl
	     << "Bandwidth after MC64: " << stats.bandwidthMC64 << endl
	     << "Bandwidth after RCM:  " << stats.bandwidthReorder << endl
	     << "Bandwidth final:      " << stats.bandwidth << endl
	     << "nuKf factor:          " << stats.nuKf << endl << endl;

	cusp::copy(anew, a);
	cusp::copy(v, vnew);
	cusp::copy(p, pnew);


	// Vectors for Spike solver stats
	spikeSolveTime.resize(maxNewtonIterations);
	spikeNumIter.resize(maxNewtonIterations);


	//ANCFSystem::updateParticleDynamics();

	return 0;
}

int ANCFSystem::DoTimeStep() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//ANCFSystem::updateParticleDynamics();
	stepKrylovIterations = 0;
	precUpdated = false;

	// update q and q_dot for initial guess
	cusp::blas::axpbypcz(p, v, a, pnew, 1, h, .5 * h * h);
	cusp::blas::axpby(v, a, vnew, 1, h);

	// Force a preconditioner update if needed
	if ((preconditionerUpdateModulus > 0) && (timeIndex % preconditionerUpdateModulus == 0)) {
		mySolver->update(lhs.values);
		precUpdated = true;
		printf("Preconditioner updated (step condition)!\n");
	}

	// Perform Newton iterations
	int it;
	for (it = 0; it < maxNewtonIterations; it++) {
		ANCFSystem::updatePhi();
		cusp::multiply(phiq, lambda, phiqlam);
		ANCFSystem::resetLeftHandSideMatrix();
		cusp::multiply(lhs_mass, anew, eTop); //cusp::multiply(mass,anew,eTop);
		ANCFSystem::updateInternalForces();
		cusp::blas::axpbypcz(eTop, fapp, fint, eTop, 1, -1, 1);
		cusp::blas::axpby(eTop, fext, eTop, 1, -1);
		cusp::blas::axpy(phiqlam, eTop, 1);
		cusp::blas::copy(phi, eBottom);

		// SOLVE THE LINEAR SYSTEM USING SPIKE
		cusp::blas::fill(delta, 0); // very important
		//stencil lhsStencil(anewAll.size(), lhs_mass, lhs_phiq, lhsVec);

		bool success = mySolver->solve(*m_spmv, eAll, delta);
		spike::Stats stats = mySolver->getStats();

		if(!success) {
			printf("**********  DUMP DATA **************\n");

			char filename[100];
			
			sprintf(filename, "./data/lhs%d.mtx", timeIndex);
			cusp::io::write_matrix_market_file(lhs, filename);

			sprintf(filename, "./data/rhs%d.mtx", timeIndex);
			cusp::io::write_matrix_market_file(eAll, filename);

			sprintf(filename, "./data/stats%d.txt", timeIndex);
			ofstream file(filename);
			file << "Code: " << mySolver->getMonitorCode();
			file << "  " << mySolver->getMonitorMessage() << std::endl;
			file << "Number of iterations = " << stats.numIterations << std::endl;
			file << "RHS norm             = " << stats.rhsNorm << std::endl;
			file << "Residual norm        = " << stats.residualNorm << std::endl;
			file << "Rel. residual norm   = " << stats.relResidualNorm << std::endl;
			file.close();

			int code = mySolver->getMonitorCode();
			if (code == -1 || code == -2) {

				//// TODO:  clean this up...

				std::cout << "STOP" << std::endl;
				exit(0);
			}
		}

		spikeSolveTime[it] = stats.timeSolve;
		spikeNumIter[it] = stats.numIterations;
		stepKrylovIterations += stats.numIterations;
		// END SOLVE THE LINEAR SYSTEM

		// update anew
		cusp::blas::axpy(delta, anewAll, -1);

		// update vnew
		cusp::blas::axpbypcz(v, a, anew, vnew, 1, h * (1 - gammaHHT), h * gammaHHT);

		// update pnew
		cusp::blas::axpbypcz(v, a, anew, pnew, h, h * h * .5 * (1 - 2 * betaHHT), h * h * .5 * 2 * betaHHT);
		cusp::blas::axpy(p, pnew, 1);

		// Calculate infinity norm of the correction and check for convergence
		double delta_nrm = cusp::blas::nrmmax(delta);

		printf("         Krylov solver: %8.2f ms    %.2f iterations     ||delta||_inf = %e\n",
			stats.timeSolve, stats.numIterations, delta_nrm);

		if (delta_nrm <= tol)
			break;
	}

	// Number of Newton iterations and average number of Krylov iterations
	stepNewtonIterations = it + 1;
	float avgKrylov = stepKrylovIterations / stepNewtonIterations;

	// If the average number of Krylov iterations per Newton iteration exceeds the specified limit,
	// force a preconditioner update.
	if ((preconditionerMaxKrylovIterations > 0) && (avgKrylov > preconditionerMaxKrylovIterations)) {
		ANCFSystem::updateInternalForces();
		mySolver->update(lhs.values);
		precUpdated = true;
		printf("Preconditioner updated! (krylov condition)\n");
	}

	cusp::copy(anew, a);
	cusp::copy(vnew, v);
	cusp::copy(pnew, p);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	stepTime = elapsedTime;
	timeToSimulate += elapsedTime / 1000.0;

	p_h = p_d;

//	char filename[100];
//	sprintf(filename, "./data/lhs%d.txt",timeIndex);
//	cusp::io::write_matrix_market_file(lhs, filename);
//
//	char filename1[100];
//	sprintf(filename1, "./data/rhs%d.txt",timeIndex);
//	cusp::io::write_matrix_market_file(eAll, filename1);

	time += h;
	timeIndex++;


	printf("%f, Elapsed time = %8.2f ms, Newton = %2d, Ave. Krylov Per Newton = %.2f\n",
	       time, elapsedTime, stepNewtonIterations, avgKrylov);


	return 0;
}

float3 ANCFSystem::getXYZPosition(int elementIndex, double xi) {
	double a = elements[elementIndex].getLength_l();
	double* p = CASTD1(p_h);
	p = &p[12 * elementIndex];
	float3 pos;

	pos.x = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[0]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[3]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[6]
			+ a * (-xi * xi + pow(xi, 3)) * p[9];
	pos.y = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[1]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[4]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[7]
			+ a * (-xi * xi + pow(xi, 3)) * p[10];
	pos.z = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[2]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[5]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[8]
			+ a * (-xi * xi + pow(xi, 3)) * p[11];

	return pos;
}

float3 ANCFSystem::getXYZVelocity(int elementIndex, double xi) {
	double a = elements[elementIndex].getLength_l();
	double* p = CASTD1(v_h);
	p = &p[12 * elementIndex];
	float3 pos;

	pos.x = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[0]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[3]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[6]
			+ a * (-xi * xi + pow(xi, 3)) * p[9];
	pos.y = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[1]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[4]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[7]
			+ a * (-xi * xi + pow(xi, 3)) * p[10];
	pos.z = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[2]
			+ a * (xi - 2 * xi * xi + pow(xi, 3)) * p[5]
			+ (3 * xi * xi - 2 * pow(xi, 3)) * p[8]
			+ a * (-xi * xi + pow(xi, 3)) * p[11];

	return pos;
}

float3 ANCFSystem::getXYZPositionParticle(int index) {
	return make_float3(pParticle_h[3 * index], pParticle_h[3 * index + 1],
			pParticle_h[3 * index + 2]);
}

float3 ANCFSystem::getXYZVelocityParticle(int index) {
	return make_float3(vParticle_h[3 * index], vParticle_h[3 * index + 1],
			vParticle_h[3 * index + 2]);
}

int ANCFSystem::saveLHS() {
	posFile.open("../lhs.dat");
	posFile << "symmetric" << endl;
	posFile << anew_h.size() << " " << anew_h.size() << " " << lhsI_h.size()
			<< endl;
	for (int i = 0; i < lhsI_h.size(); i++) {
		posFile << lhsI_h[i] << " " << lhsJ_h[i] << " " << lhs_h[i] << endl;
	}
	posFile.close();

	return 0;
}

int ANCFSystem::writeToFile(string fileName) {
	//char filename1[100];
	//sprintf(filename1, "./posData/lhs%d.dat", fileIndex);
	//cusp::io::write_matrix_market_file(lhs, filename1);

	posFile.open(fileName.c_str());
	p_h = p_d;
	double* posAll = CASTD1(p_h);
	double* pos;
	double l;
	double r;
	posFile << elements.size() << "," << endl;
//  float3 posPart;
//	for(int i=0;i<particles.size();i++)
//	{
//		r = particles[i].getRadius();
//		posPart = getXYZPositionParticle(i);
//		posFile << r << ", " << posPart.x << ", " << posPart.y << ", " << posPart.z << "," << endl;
//	}
	for (int i = 0; i < elements.size(); i++) {
		l = elements[i].getLength_l();
		r = elements[i].getRadius();
		pos = &posAll[12 * i];
		posFile << r << "," << l;
		for (int i = 0; i < 12; i++)
			posFile << "," << pos[i];
		posFile << "," << endl;
	}
	posFile.close();

	return 0;
}
