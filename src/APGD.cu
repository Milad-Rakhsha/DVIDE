#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "APGD.cuh"

APGD::APGD(System* sys)
{
  system = sys;

  tolerance = 1e-4;
  maxIterations = 100000;
  iterations = 0;
}

int APGD::setup()
{
  gammaHat_d = system->a_h;
  gammaNew_d = system->a_h;
  g_d = system->a_h;
  y_d = system->a_h;
  yNew_d = system->a_h;
  gammaTmp_d = system->a_h;

  thrust::device_ptr<double> wrapped_device_gammaHat(CASTD1(gammaHat_d));
  thrust::device_ptr<double> wrapped_device_gammaNew(CASTD1(gammaNew_d));
  thrust::device_ptr<double> wrapped_device_g(CASTD1(g_d));
  thrust::device_ptr<double> wrapped_device_y(CASTD1(y_d));
  thrust::device_ptr<double> wrapped_device_yNew(CASTD1(yNew_d));
  thrust::device_ptr<double> wrapped_device_gammaTmp(CASTD1(gammaTmp_d));

  gammaHat = DeviceValueArrayView(wrapped_device_gammaHat, wrapped_device_gammaHat + gammaHat_d.size());
  gammaNew = DeviceValueArrayView(wrapped_device_gammaNew, wrapped_device_gammaNew + gammaNew_d.size());
  g = DeviceValueArrayView(wrapped_device_g, wrapped_device_g + g_d.size());
  y = DeviceValueArrayView(wrapped_device_y, wrapped_device_y + y_d.size());
  yNew = DeviceValueArrayView(wrapped_device_yNew, wrapped_device_yNew + yNew_d.size());
  gammaTmp = DeviceValueArrayView(wrapped_device_gammaTmp, wrapped_device_gammaTmp + gammaTmp_d.size());

  return 0;
}

__global__ void project(double* src, double* friction, uint offsetBilateralConstraints, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double mu = friction[index]; // TODO: Keep an eye on friction indexing
  src = &src[3*index+offsetBilateralConstraints];
  double3 gamma = make_double3(src[0],src[1],src[2]);
  double gamma_n = gamma.x;
  double gamma_t = sqrt(pow(gamma.y,2.0)+pow(gamma.z,2.0));

  if(mu == 0) {
    gamma = make_double3(gamma_n,0,0);
    if (gamma_n < 0) gamma = make_double3(0,0,0);
  }
  else if(gamma_t < mu * gamma_n) {
    // Don't touch gamma!
  }
  else if((gamma_t < -(1.0/mu)*gamma_n) || (abs(gamma_n) < 10e-15)) {
    gamma = make_double3(0,0,0);
  }
  else {
    double gamma_n_proj = (gamma_t * mu + gamma_n)/(pow(mu,2.0)+1.0);
    double gamma_t_proj = gamma_n_proj * mu;
    double tproj_div_t = gamma_t_proj/gamma_t;
    double gamma_u_proj = tproj_div_t * gamma.y;
    double gamma_v_proj = tproj_div_t * gamma.z;
    gamma = make_double3(gamma_n_proj, gamma_u_proj, gamma_v_proj);
  }

  src[0] = gamma.x;
  src[1] = gamma.y;
  src[2] = gamma.z;
}


int APGD::performSchurComplementProduct(DeviceValueArrayView src) {
  cusp::multiply(system->DT,src,system->f_contact);
  cusp::multiply(system->mass,system->f_contact,system->tmp);
  cusp::multiply(system->D,system->tmp,gammaTmp);

  return 0;
}


double APGD::getResidual(DeviceValueArrayView src) {
  double gdiff = 1.0 / pow(system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size(),2.0);
  performSchurComplementProduct(src); //cusp::multiply(system->N,src,gammaTmp); //
  cusp::blas::axpy(system->r,gammaTmp,1.0);
  cusp::blas::axpby(src,gammaTmp,gammaTmp,1.0,-gdiff);
  if(system->collisionDetector->numCollisions) project<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaTmp_d), CASTD1(system->friction_d), system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), system->collisionDetector->numCollisions);
  cusp::blas::axpby(src,gammaTmp,gammaTmp,1.0/gdiff,-1.0/gdiff);

  return cusp::blas::nrmmax(gammaTmp);
}

__global__ void updateAntiRelaxationVector(double* s, double* friction, double* antiRelaxation, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double s_v = s[3*index+1];
  double s_w = s[3*index+2];
  double mu = friction[index]; // TODO: Keep an eye on friction indexing

  antiRelaxation[3*index] = sqrt(pow(s_v,2.0)+pow(s_w,2.0))*mu;
  antiRelaxation[3*index+1] = 0;
  antiRelaxation[3*index+2] = 0;
}

__global__ void initializeImpulseVector_APGD(double* src, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  src[3*index  ] = 1.0;
  src[3*index+1] = 0.0;
  src[3*index+2] = 0.0;
}

__global__ void getResidual_APGD(double* src, double* gamma, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  src[3*index] = src[3*index]*gamma[3*index]+src[3*index+1]*gamma[3*index+1]+src[3*index+2]*gamma[3*index+2];
  src[3*index+1] = 0;
  src[3*index+2] = 0;
}

__global__ void getFeasibleX_APGD(double* src, double* dst, double* friction, uint numBilaterals, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double mu = friction[index]; // TODO: Keep an eye on friction indexing

  double xn = src[3*index+numBilaterals];
  double xt1 = src[3*index+1+numBilaterals];
  double xt2 = src[3*index+2+numBilaterals];

  xn = mu*xn-sqrt(pow(xt1,2.0)+pow(xt2,2.0));
  if(xn!=xn) xn = 0.0;
  dst[3*index+numBilaterals] = -fmin(0.0,xn);
  dst[3*index+1+numBilaterals] = -10e30;
  dst[3*index+2+numBilaterals] = -10e30;
}

__global__ void getFeasibleY_APGD(double* src, double* dst, double* friction, uint numBilaterals, uint numCollisions) {
  INIT_CHECK_THREAD_BOUNDED(INDEX1D, numCollisions);

  double mu = friction[index]; // TODO: Keep an eye on friction indexing

  double xn = src[3*index+numBilaterals];
  double xt1 = src[3*index+1+numBilaterals];
  double xt2 = src[3*index+2+numBilaterals];

  xn = (1.0/mu)*xn-sqrt(pow(xt1,2.0)+pow(xt2,2.0));
  if(xn!=xn) xn = 0.0;
  dst[3*index+numBilaterals] = -fmin(0.0,xn);
  dst[3*index+1+numBilaterals] = -10e30;
  dst[3*index+2+numBilaterals] = -10e30;
}

int APGD::solve() {

  system->gamma_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  gammaHat_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  gammaNew_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  g_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  y_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  yNew_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  gammaTmp_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());
  antiRelaxation_d.resize(3*system->collisionDetector->numCollisions+system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size());

  // TODO: There's got to be a better way to do this...
  thrust::device_ptr<double> wrapped_device_gamma(CASTD1(system->gamma_d));
  thrust::device_ptr<double> wrapped_device_gammaHat(CASTD1(gammaHat_d));
  thrust::device_ptr<double> wrapped_device_gammaNew(CASTD1(gammaNew_d));
  thrust::device_ptr<double> wrapped_device_g(CASTD1(g_d));
  thrust::device_ptr<double> wrapped_device_y(CASTD1(y_d));
  thrust::device_ptr<double> wrapped_device_yNew(CASTD1(yNew_d));
  thrust::device_ptr<double> wrapped_device_gammaTmp(CASTD1(gammaTmp_d));
  thrust::device_ptr<double> wrapped_device_antiRelaxation(CASTD1(antiRelaxation_d));
  system->gamma = DeviceValueArrayView(wrapped_device_gamma, wrapped_device_gamma + system->gamma_d.size());
  gammaHat = DeviceValueArrayView(wrapped_device_gammaHat, wrapped_device_gammaHat + gammaHat_d.size());
  gammaNew = DeviceValueArrayView(wrapped_device_gammaNew, wrapped_device_gammaNew + gammaNew_d.size());
  g = DeviceValueArrayView(wrapped_device_g, wrapped_device_g + g_d.size());
  y = DeviceValueArrayView(wrapped_device_y, wrapped_device_y + y_d.size());
  yNew = DeviceValueArrayView(wrapped_device_yNew, wrapped_device_yNew + yNew_d.size());
  gammaTmp = DeviceValueArrayView(wrapped_device_gammaTmp, wrapped_device_gammaTmp + gammaTmp_d.size());
  antiRelaxation = DeviceValueArrayView(wrapped_device_antiRelaxation, wrapped_device_antiRelaxation + antiRelaxation_d.size());

  // (1) gamma_0 = zeros(nc,1)
  cusp::blas::fill(antiRelaxation,0.0);
  //cusp::blas::fill(system->gamma,0);

  // Provide an initial guess for gamma
  //initializeImpulseVector_APGD<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(system->gamma_d), system->collisionDetector->numCollisions);

  // (2) gamma_hat_0 = ones(nc,1)
  cusp::blas::fill(gammaHat,1.0);

  // (3) y_0 = gamma_0
  cusp::blas::copy(system->gamma,y);

  // (4) theta_0 = 1
  double theta = 1.0;
  double thetaNew = theta;
  double Beta = 0.0;
  double obj1 = 0.0;
  double obj2 = 0.0;
  double residual = 10e30;

  // (5) L_k = norm(N * (gamma_0 - gamma_hat_0)) / norm(gamma_0 - gamma_hat_0)
  cusp::blas::axpby(system->gamma,gammaHat,gammaTmp,1.0,-1.0);
  double L = cusp::blas::nrm2(gammaTmp);
  performSchurComplementProduct(gammaTmp); //cusp::multiply(system->N,gammaTmp,g); //
  L = cusp::blas::nrm2(gammaTmp)/L;

  // (6) t_k = 1 / L_k
  double t = 1.0/L;

  // (7) for k := 0 to N_max
  int k;
  for (k=0; k < maxIterations; k++) {
    // (8) g = N * y_k - r
    performSchurComplementProduct(y); //cusp::multiply(system->N,y,gammaTmp); //
    cusp::blas::axpby(gammaTmp,system->r,g,1.0,1.0);

    // (9) gamma_(k+1) = ProjectionOperator(y_k - t_k * g)
    cusp::blas::axpby(y,g,gammaNew,1.0,-t);
    if(system->collisionDetector->numCollisions) project<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaNew_d), CASTD1(system->friction_d), system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), system->collisionDetector->numCollisions);

    // (10) while 0.5 * gamma_(k+1)' * N * gamma_(k+1) - gamma_(k+1)' * r >= 0.5 * y_k' * N * y_k - y_k' * r + g' * (gamma_(k+1) - y_k) + 0.5 * L_k * norm(gamma_(k+1) - y_k)^2
    performSchurComplementProduct(gammaNew); //cusp::multiply(system->N,gammaNew,gammaTmp); //
    obj1 = 0.5 * cusp::blas::dot(gammaNew,gammaTmp) + cusp::blas::dot(gammaNew,system->r);
    performSchurComplementProduct(y); //cusp::multiply(system->N,y,gammaTmp); //
    obj2 = 0.5 * cusp::blas::dot(y,gammaTmp) + cusp::blas::dot(y,system->r);
    cusp::blas::axpby(gammaNew,y,gammaTmp,1.0,-1.0);
    obj2 += cusp::blas::dot(g,gammaTmp) + 0.5 * L * pow(cusp::blas::nrm2(gammaTmp),2.0);

    while (obj1 >= obj2) {
      // (11) L_k = 2 * L_k
      L = 2.0 * L;

      // (12) t_k = 1 / L_k
      t = 1.0 / L;

      // (13) gamma_(k+1) = ProjectionOperator(y_k - t_k * g)
      cusp::blas::axpby(y,g,gammaNew,1.0,-t);
      if(system->collisionDetector->numCollisions) project<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaNew_d), CASTD1(system->friction_d), system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), system->collisionDetector->numCollisions);

      // Update the components of the while condition
      performSchurComplementProduct(gammaNew); //cusp::multiply(system->N,gammaNew,gammaTmp); //
      obj1 = 0.5 * cusp::blas::dot(gammaNew,gammaTmp) + cusp::blas::dot(gammaNew,system->r);
      performSchurComplementProduct(y); //cusp::multiply(system->N,y,gammaTmp); //
      obj2 = 0.5 * cusp::blas::dot(y,gammaTmp) + cusp::blas::dot(y,system->r);
      cusp::blas::axpby(gammaNew,y,gammaTmp,1.0,-1.0);
      obj2 += cusp::blas::dot(g,gammaTmp) + 0.5 * L * pow(cusp::blas::nrm2(gammaTmp),2.0);

      // (14) endwhile
    }

    // (15) theta_(k+1) = (-theta_k^2 + theta_k * sqrt(theta_k^2 + 4)) / 2
    thetaNew = (-pow(theta, 2.0) + theta * sqrt(pow(theta, 2.0) + 4.0)) / 2.0;

    // (16) Beta_(k+1) = theta_k * (1 - theta_k) / (theta_k^2 + theta_(k+1))
    Beta = theta * (1.0 - theta) / (pow(theta, 2.0) + thetaNew);

    // (17) y_(k+1) = gamma_(k+1) + Beta_(k+1) * (gamma_(k+1) - gamma_k)
    cusp::blas::axpby(gammaNew,system->gamma,yNew,(1.0+Beta),-Beta);

    // (18) r = r(gamma_(k+1))
    double res = getResidual(gammaNew);
//    getFeasibleX_APGD<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaNew_d), CASTD1(gammaTmp_d), CASTD1(system->friction_d), system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), system->collisionDetector->numCollisions);
//    thrust::fill(gammaTmp_d.begin(), gammaTmp_d.begin() + system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), -10e30);
//    double feasibleX = Thrust_Max(gammaTmp_d);
//
//    performSchurComplementProduct(gammaNew);
//    cusp::blas::axpy(system->r,gammaTmp,1.0);
//    double res3 = abs(cusp::blas::dot(gammaNew,gammaTmp));
//
//    getFeasibleY_APGD<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaTmp_d), CASTD1(gammaTmp_d), CASTD1(system->friction_d), system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), system->collisionDetector->numCollisions);
//    thrust::fill(gammaTmp_d.begin(), gammaTmp_d.begin() + system->constraintsBilateralDOF_d.size()+3*system->constraintsSpherical_ShellNodeToBody2D_d.size(), -10e30);
//    double feasibleY = Thrust_Max(gammaTmp_d);
//
//    double res = fmax(feasibleX,feasibleY);
//    res = fmax(res,res3);

    // (19) if r < epsilon_min
    if (res < residual) {
      // (20) r_min = r
      residual = res;

      // (21) gamma_hat = gamma_(k+1)
      cusp::blas::copy(gammaNew,gammaHat);

      // (22) endif
    }

    // (23) if r < Tau
    if (residual < tolerance) {
      // (24) break
      break;

      // (25) endif
    }

    // (26) if g' * (gamma_(k+1) - gamma_k) > 0
    cusp::blas::axpby(gammaNew,system->gamma,gammaTmp,1.0,-1.0);
    if (cusp::blas::dot(g,gammaTmp) > 0) {
      // (27) y_(k+1) = gamma_(k+1)
      cusp::blas::copy(gammaNew,yNew);

      // (28) theta_(k+1) = 1
      thetaNew = 1.0;

      // (29) endif
    }

    // (30) L_k = 0.9 * L_k
    L = 0.9 * L;

    // (31) t_k = 1 / L_k
    t = 1.0 / L;

    // Update iterates
    theta = thetaNew;
    cusp::blas::copy(gammaNew,system->gamma);
    cusp::blas::copy(yNew,y);

//    // Apply anti-relaxation
//    cusp::blas::axpy(antiRelaxation,system->r,-1.0);
//    cusp::multiply(system->DT,system->gamma,system->f_contact);
//    cusp::blas::axpby(system->k,system->f_contact,system->tmp,1.0,1.0);
//    cusp::multiply(system->mass,system->tmp,system->v);
//    cusp::multiply(system->D,system->v,gammaTmp);
//    updateAntiRelaxationVector<<<BLOCKS(system->collisionDetector->numCollisions),THREADS>>>(CASTD1(gammaTmp), CASTD1(system->friction_d), CASTD1(antiRelaxation), system->collisionDetector->numCollisions);
//    cusp::blas::axpy(antiRelaxation,system->r,1.0);
//    // End apply anti-relaxation

    // (32) endfor
    //cout << "  Iterations: " << k << " Residual: " << residual << endl;
  }
  cout << "  Iterations: " << k << " Residual: " << residual << endl;

  // (33) return Value at time step t_(l+1), gamma_(l+1) := gamma_hat
  iterations = k;
  cusp::blas::copy(gammaHat,system->gamma);

  return 0;
}
