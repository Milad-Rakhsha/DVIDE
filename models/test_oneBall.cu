#include "hip/hip_runtime.h"
#include "include.cuh"
#include <sys/stat.h>
#include <errno.h>
#include "System.cuh"
#include "Body.cuh"
#include "APGD.cuh"
#include "PDIP.cuh"
#include "TPAS.cuh"
#include "JKIP.cuh"

bool updateDraw = 1;
bool wireFrame = 1;

// Create the system (placed outside of main so it is available to the OpenGL code)
System* sys;
std::string outDir = "../TEST_ONEBALL/";
std::string povrayDir = outDir + "POVRAY/";

#ifdef WITH_GLUT
OpenGLCamera oglcamera(camreal3(0,0,-3),camreal3(0,0,0),camreal3(0,1,0),.01);

// OPENGL RENDERING CODE //
void changeSize(int w, int h) {
	if(h == 0) {h = 1;}
	float ratio = 1.0* w / h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, w, h);
	gluPerspective(45,ratio,.1,1000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(0.0,0.0,0.0,		0.0,0.0,-7,		0.0f,1.0f,0.0f);
}

void initScene(){
	GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };
	glClearColor (1.0, 1.0, 1.0, 0.0);
	glShadeModel (GL_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable (GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glHint (GL_POINT_SMOOTH_HINT, GL_DONT_CARE);
}

void drawAll()
{
	if(updateDraw){
	  sys->p_h = sys->p_d;
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glEnable(GL_DEPTH_TEST);
		glFrontFace(GL_CCW);
		glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
		glDepthFunc(GL_LEQUAL);
		glClearDepth(1.0);

		glPointSize(2);
		glLoadIdentity();

		oglcamera.Update();

		for(int i=0;i<sys->bodies.size();i++)
		{
			if(wireFrame) {
			  glPushMatrix();
			  double3 position = sys->bodies[i]->getPosition();
			  glTranslatef(sys->p_h[3*i],sys->p_h[3*i+1],sys->p_h[3*i+2]);
			  double3 geometry = sys->bodies[i]->getGeometry();
			  if(geometry.y) {
			    glColor3f(0.0f,1.0f,0.0f);
			    glScalef(2*geometry.x, 2*geometry.y, 2*geometry.z);
			    glutWireCube(1.0);
			  }
			  else {
			    glColor3f(0.0f,0.0f,1.0f);
			    glutWireSphere(geometry.x,30,30);
			  }
			  glPopMatrix();
			}
			else {
        glPushMatrix();
        double3 position = sys->bodies[i]->getPosition();
        glTranslatef(sys->p_h[3*i],sys->p_h[3*i+1],sys->p_h[3*i+2]);
        double3 geometry = sys->bodies[i]->getGeometry();
        if(geometry.y) {
          glColor3f(0.0f,1.0f,0.0f);
          glScalef(2*geometry.x, 2*geometry.y, 2*geometry.z);
          glutSolidCube(1.0);
        }
        else {
          glColor3f(0.0f,0.0f,1.0f);
          glutSolidSphere(geometry.x,30,30);
        }
        glPopMatrix();
      }
		}

		glutSwapBuffers();
	}
}

void renderSceneAll(){
	if(OGL){
		//if(sys->timeIndex%10==0)
		drawAll();
    sys->DoTimeStep();
    double4 violation = sys->getCCPViolation();
    printf("  Violation: (%f, %f, %f, %f)\n", violation.x, violation.y, violation.z, violation.w);

    // Determine contact force on the ball
    sys->f_contact_h = sys->f_contact_d;
    cout << "  fNorm: " << sys->f_contact_h[1] << endl;
    cout << "  fTang: " << sys->f_contact_h[0] << endl;
    cin.get();
	}
}

void CallBackKeyboardFunc(unsigned char key, int x, int y) {
	switch (key) {
	case 'w':
		oglcamera.Forward();
		break;

	case 's':
		oglcamera.Back();
		break;

	case 'd':
		oglcamera.Right();
		break;

	case 'a':
		oglcamera.Left();
		break;

	case 'q':
		oglcamera.Up();
		break;

	case 'e':
		oglcamera.Down();
		break;

	case 'i':
	  if(wireFrame) {
	    wireFrame = 0;
	  }
	  else {
	    wireFrame = 1;
	  }
	}
}

void CallBackMouseFunc(int button, int state, int x, int y) {
	oglcamera.SetPos(button, state, x, y);
}
void CallBackMotionFunc(int x, int y) {
	oglcamera.Move2D(x, y);
}
#endif
// END OPENGL RENDERING CODE //

double getRandomNumber(double min, double max)
{
  // x is in [0,1[
  double x = rand()/static_cast<double>(RAND_MAX);

  // [0,1[ * (max - min) + min is in [min,max[
  double that = min + ( x * (max - min) );

  return that;
}

int main(int argc, char** argv)
{
	// command line arguments
	// FlexibleNet <numPartitions> <numBeamsPerSide> <solverType> <usePreconditioning>
	// solverType: (0) BiCGStab, (1) BiCGStab1, (2) BiCGStab2, (3) MinRes, (4) CG, (5) CR

  double t_end = 5.0;
  int    precUpdateInterval = -1;
  float  precMaxKrylov = -1;
  int precondType = 1;
  double latForce = 0;
  int solverType = 4;
  int numPartitions = 1;
  double mu_pdip = 10.0;
  double alpha = 0.01; // should be [0.01, 0.1]
  double beta = 0.8; // should be [0.3, 0.8]
  int solverTypeQOCC = 1;
  int binsPerAxis = 10;
  double tolerance = 1e-5;
  double hh = 1e-2;

  if(argc > 1) {
    latForce = atof(argv[1]);
    solverTypeQOCC = atoi(argv[2]);
    tolerance = atof(argv[3]);
  }

#ifdef WITH_GLUT
	bool visualize = true;
#endif
	//visualize = false;

	sys = new System(solverTypeQOCC);
  sys->setTimeStep(hh);
  sys->solver->tolerance = tolerance;
  sys->gravity = make_double3(latForce,-9.81,0);

  sys->collisionDetector->setBinsPerAxis(make_uint3(binsPerAxis,binsPerAxis,binsPerAxis));
  if(solverTypeQOCC==1) {
    dynamic_cast<APGD*>(sys->solver)->setWarmStarting(false);
    dynamic_cast<APGD*>(sys->solver)->setAntiRelaxation(false);
  }
  if(solverTypeQOCC==2) {
    dynamic_cast<PDIP*>(sys->solver)->setPrecondType(precondType);
    dynamic_cast<PDIP*>(sys->solver)->setSolverType(solverType);
    dynamic_cast<PDIP*>(sys->solver)->setNumPartitions(numPartitions);
    dynamic_cast<PDIP*>(sys->solver)->alpha = alpha;
    dynamic_cast<PDIP*>(sys->solver)->beta = beta;
    dynamic_cast<PDIP*>(sys->solver)->mu_pdip = mu_pdip;
  }
  if(solverTypeQOCC==3) {
    dynamic_cast<TPAS*>(sys->solver)->setPrecondType(precondType);
    dynamic_cast<TPAS*>(sys->solver)->setSolverType(solverType);
    dynamic_cast<TPAS*>(sys->solver)->setNumPartitions(numPartitions);
    dynamic_cast<TPAS*>(sys->solver)->alpha = alpha;
    dynamic_cast<TPAS*>(sys->solver)->beta = beta;
    dynamic_cast<TPAS*>(sys->solver)->mu_pdip = mu_pdip;
  }
  if(solverTypeQOCC==4) {
    dynamic_cast<JKIP*>(sys->solver)->setPrecondType(precondType);
    dynamic_cast<JKIP*>(sys->solver)->setSolverType(solverType);
    dynamic_cast<JKIP*>(sys->solver)->setNumPartitions(numPartitions);
    dynamic_cast<JKIP*>(sys->solver)->careful = true;
  }

  // Bottom
  Body* ground = new Body(make_double3(0,-1,0));
  ground->setBodyFixed(true);
  ground->setGeometry(make_double3(1,1,1));
  sys->add(ground);

  Body* ball = new Body(make_double3(0,1,0));
  ball->setGeometry(make_double3(1,0,0));
  //ball1->setMass(20);
  sys->add(ball);

	sys->initializeSystem();
	printf("System initialized!\n");

#ifdef WITH_GLUT
	if(visualize)
	{
		glutInit(&argc, argv);
		glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
		glutInitWindowPosition(0,0);
		glutInitWindowSize(1024	,512);
		glutCreateWindow("MAIN");
		glutDisplayFunc(renderSceneAll);
		glutIdleFunc(renderSceneAll);
		glutReshapeFunc(changeSize);
		glutIgnoreKeyRepeat(0);
		glutKeyboardFunc(CallBackKeyboardFunc);
		glutMouseFunc(CallBackMouseFunc);
		glutMotionFunc(CallBackMotionFunc);
		initScene();
		glutMainLoop();
	}
#endif

	while(sys->time < t_end)
	{
		sys->DoTimeStep();
		double4 violation = sys->getCCPViolation();
		printf("  Violation: (%f, %f, %f, %f)\n", violation.x, violation.y, violation.z, violation.w);

		// Determine contact force on the ball
		sys->f_contact_h = sys->f_contact_d;
		cout << "  fNorm: " << sys->f_contact_h[1] << endl;
		cout << "  fTang: " << sys->f_contact_h[0] << endl;

		int numKrylovIter = 0;
		if(solverTypeQOCC==2) numKrylovIter = dynamic_cast<PDIP*>(sys->solver)->totalKrylovIterations;
		if(solverTypeQOCC==3) numKrylovIter = dynamic_cast<TPAS*>(sys->solver)->totalKrylovIterations;
		if(solverTypeQOCC==4) numKrylovIter = dynamic_cast<JKIP*>(sys->solver)->totalKrylovIterations;
	}

	return 0;
}

