#include "hip/hip_runtime.h"
#include "include.cuh"
#include "System.cuh"
#include "Element.cuh"

bool updateDraw = 1;
bool showSphere = 1;

// Create the system (placed outside of main so it is available to the OpenGL code)
System sys;

#ifdef WITH_GLUT
OpenGLCamera oglcamera(camreal3(-1,1,-1),camreal3(0,0,0),camreal3(0,1,0),.01);

// OPENGL RENDERING CODE //
void changeSize(int w, int h) {
	if(h == 0) {h = 1;}
	float ratio = 1.0* w / h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, w, h);
	gluPerspective(45,ratio,.1,1000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(0.0,0.0,0.0,		0.0,0.0,-7,		0.0f,1.0f,0.0f);
}

void initScene(){
	GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };
	glClearColor (1.0, 1.0, 1.0, 0.0);
	glShadeModel (GL_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable (GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glHint (GL_POINT_SMOOTH_HINT, GL_DONT_CARE);
}

void drawAll()
{
	if(updateDraw){
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glEnable(GL_DEPTH_TEST);
		glFrontFace(GL_CCW);
		glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
		glDepthFunc(GL_LEQUAL);
		glClearDepth(1.0);

		glPointSize(2);
		glLoadIdentity();

		oglcamera.Update();

		for(int i=0;i<sys.elements.size();i++)
		{
			//if(showSphere)
			{
				glColor3f(0.0f,0.0f,1.0f);
				//for(int j=0;j<xiDiv;j++)
				{
					glPushMatrix();
					double3 position = sys.elements[i].getPosition();
					glTranslatef(sys.p_h[3*i],sys.p_h[3*i+1],sys.p_h[3*i+2]);
					glutSolidSphere(1,10,10);
					glPopMatrix();
				}
			}
		}

		glutSwapBuffers();
	}
}

void renderSceneAll(){
	if(OGL){
		//if(sys.timeIndex%10==0)
		drawAll();
		sys.DoTimeStep();
	}
}

void CallBackKeyboardFunc(unsigned char key, int x, int y) {
	switch (key) {
	case 'w':
		oglcamera.Forward();
		break;
	case 's':
		oglcamera.Back();
		break;

	case 'd':
		oglcamera.Right();
		break;

	case 'a':
		oglcamera.Left();
		break;

	case 'q':
		oglcamera.Up();
		break;

	case 'e':
		oglcamera.Down();
		break;
	}
}

void CallBackMouseFunc(int button, int state, int x, int y) {
	oglcamera.SetPos(button, state, x, y);
}
void CallBackMotionFunc(int x, int y) {
	oglcamera.Move2D(x, y);
}
#endif
// END OPENGL RENDERING CODE //

int main(int argc, char** argv)
{
	// command line arguments
	// ImplicitBeamsGPU <numPartitions> <numBeamsPerSide> <solverType> <usePreconditioning> <elasticModulus> <dataFolder>
	// solverType: (0) BiCGStab, (1) BiCGStab1, (2) BiCGStab2, (3) MinRes

#ifdef WITH_GLUT
	bool visualize = true;
#endif

  sys.setTimeStep(1e-3, 1e-10);
  sys.setMaxKrylovIterations(5000);
  double t_end = 5.0;
  int    precUpdateInterval = -1;
  float  precMaxKrylov = -1;

	sys.setNumPartitions((int)atoi(argv[1]));
  int numElementsPerSide = atoi(argv[2]);
  sys.setSolverType((int)atoi(argv[3]));
  sys.setPrecondType(atoi(argv[4]));
  if(atoi(argv[4])) {
    sys.preconditionerUpdateModulus = precUpdateInterval;
    sys.preconditionerMaxKrylovIterations = precMaxKrylov;
  }

	Element element;
	int k = 0;
	// Add elements in x-direction
	for (int j = 0; j < numElementsPerSide+1; j++) {
	  for (int i = 0; i < numElementsPerSide; i++) {
	    element = Element(make_double3(i,0,j));
	    sys.add(&element);
	    k++;
	    if(k%100==0) printf("Elements %d\n",k);
	  }
	}

	sys.initializeSystem();
	printf("System initialized!\n");
	sys.printSolverParams();
	
#ifdef WITH_GLUT
	if(visualize)
	{
		glutInit(&argc, argv);
		glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
		glutInitWindowPosition(0,0);
		glutInitWindowSize(1024	,512);
		glutCreateWindow("MAIN");
		glutDisplayFunc(renderSceneAll);
		glutIdleFunc(renderSceneAll);
		glutReshapeFunc(changeSize);
		glutIgnoreKeyRepeat(0);
		glutKeyboardFunc(CallBackKeyboardFunc);
		glutMouseFunc(CallBackMouseFunc);
		glutMotionFunc(CallBackMotionFunc);
		initScene();
		glutMainLoop();
	}
#endif
	
	// if you don't want to visualize, then output the data
	while(sys.time < t_end)
	{
		sys.DoTimeStep();
	}

	return 0;
}

