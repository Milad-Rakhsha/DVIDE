#include "hip/hip_runtime.h"
#include "include.cuh"
#include "System.cuh"
#include "Element.cuh"
#include "Node.cuh"
#include "Particle.cuh"

bool updateDraw = 1;
bool showSphere = 1;

// Create the system (placed outside of main so it is available to the OpenGL code)
System sys;

#ifdef WITH_GLUT
OpenGLCamera oglcamera(camreal3(-1,1,-1),camreal3(0,0,0),camreal3(0,1,0),.01);

// OPENGL RENDERING CODE //
void changeSize(int w, int h) {
	if(h == 0) {h = 1;}
	float ratio = 1.0* w / h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, w, h);
	gluPerspective(45,ratio,.1,1000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(0.0,0.0,0.0,		0.0,0.0,-7,		0.0f,1.0f,0.0f);
}

void initScene(){
	GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };
	glClearColor (1.0, 1.0, 1.0, 0.0);
	glShadeModel (GL_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable (GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glHint (GL_POINT_SMOOTH_HINT, GL_DONT_CARE);
}

void drawAll()
{
	if(updateDraw){
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glEnable(GL_DEPTH_TEST);
		glFrontFace(GL_CCW);
		glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
		glDepthFunc(GL_LEQUAL);
		glClearDepth(1.0);

		glPointSize(2);
		glLoadIdentity();

		oglcamera.Update();

		for(int i=0;i<sys.elements.size();i++)
		{
			int xiDiv = sys.numContactPoints;

			double xiInc = 1/(static_cast<double>(xiDiv-1));

			//if(showSphere)
			{
				glColor3f(0.0f,0.0f,1.0f);
				//for(int j=0;j<xiDiv;j++)
				{
					glPushMatrix();
					double3 position = sys.elements[i].getPosition();
					glTranslatef(sys.p_h[3*i],sys.p_h[3*i+1],sys.p_h[3*i+2]);
					glutSolidSphere(1,10,10);
					glPopMatrix();
				}
			}
//			else
//			{
//				int xiDiv = sys.numContactPoints;
//				double xiInc = 1/(static_cast<double>(xiDiv-1));
//				glLineWidth(sys.elements[i].getRadius()*500);
//				glColor3f(0.0f,1.0f,0.0f);
//				glBegin(GL_LINE_STRIP);
//				for(int j=0;j<sys.numContactPoints;j++)
//				{
//					float3 position = sys.getXYZPosition(i,xiInc*j);
//					glVertex3f(position.x,position.y,position.z);
//				}
//				glEnd();
//				glFlush();
//			}
		}

		glutSwapBuffers();
	}
}

void renderSceneAll(){
	if(OGL){
		//if(sys.timeIndex%10==0)
			drawAll();
		sys.DoTimeStep();
	}
}

void CallBackKeyboardFunc(unsigned char key, int x, int y) {
	switch (key) {
	case 'w':
		oglcamera.Forward();
		break;
	case 's':
		oglcamera.Back();
		break;

	case 'd':
		oglcamera.Right();
		break;

	case 'a':
		oglcamera.Left();
		break;

	case 'q':
		oglcamera.Up();
		break;

	case 'e':
		oglcamera.Down();
		break;
	}
}

void CallBackMouseFunc(int button, int state, int x, int y) {
	oglcamera.SetPos(button, state, x, y);
}
void CallBackMotionFunc(int x, int y) {
	oglcamera.Move2D(x, y);
}
#endif
// END OPENGL RENDERING CODE //

int main(int argc, char** argv)
{
	// command line arguments
	// ImplicitBeamsGPU <numPartitions> <numBeamsPerSide> <solverType> <usePreconditioning> <elasticModulus> <dataFolder>
	// solverType: (0) BiCGStab, (1) BiCGStab1, (2) BiCGStab2, (3) MinRes

#ifdef WITH_GLUT
	bool visualize = true;
#endif

	sys.setTimeStep(1e-3, 1e-10);
	sys.setMaxNewtonIterations(20);
	sys.setMaxKrylovIterations(5000);
	sys.setNumPartitions((int)atoi(argv[1]));
	sys.numContactPoints = 30;

	double t_end = 5.0;
	int    precUpdateInterval = -1;
	float  precMaxKrylov = -1;
	int    outputInterval = 100;

	string data_folder;

	sys.fullJacobian = 1;
	double length = 1;
	double r = .02;
	double E = 2e11;
	double rho = 2200;
	double nu = .3;
	int numElementsPerSide = atoi(argv[2]);
	sys.setSolverType((int)atoi(argv[3]));
	sys.setPrecondType(atoi(argv[4]));
	if(atoi(argv[4])) {
	  sys.preconditionerUpdateModulus = precUpdateInterval;
	  sys.preconditionerMaxKrylovIterations = precMaxKrylov;
	}
	E = atof(argv[5]);
	data_folder = argv[6];

	Element element;
	int k = 0;
	// Add elements in x-direction
	for (int j = 0; j < numElementsPerSide+1; j++) {
	  for (int i = 0; i < numElementsPerSide; i++) {
	    element = Element(make_double3(i,0,j));
	    sys.add(&element);
	    k++;
	    if(k%100==0) printf("Elements %d\n",k);
	  }
	}

	printf("%d, %d, %d\n",sys.elements.size(),sys.constraints.size(),12*sys.elements.size()+sys.constraints.size());
	sys.initializeSystem();
	printf("System initialized!\n");
	sys.printSolverParams();
	
#ifdef WITH_GLUT
	if(visualize)
	{
		glutInit(&argc, argv);
		glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
		glutInitWindowPosition(0,0);
		glutInitWindowSize(1024	,512);
		glutCreateWindow("MAIN");
		glutDisplayFunc(renderSceneAll);
		glutIdleFunc(renderSceneAll);
		glutReshapeFunc(changeSize);
		glutIgnoreKeyRepeat(0);
		glutKeyboardFunc(CallBackKeyboardFunc);
		glutMouseFunc(CallBackMouseFunc);
		glutMotionFunc(CallBackMotionFunc);
		initScene();
		glutMainLoop();
	}
#endif

	stringstream ss_m;
	ss_m << data_folder << "/" << "timing_" << atoi(argv[1]) << "_" << atoi(argv[2]) << "_" << atoi(argv[3]) << "_" << atoi(argv[4]) << "_" << atof(argv[5]) << ".txt";
	string timing_file_name = ss_m.str();
	ofstream ofile(timing_file_name.c_str());
	
	// if you don't want to visualize, then output the data
	int fileIndex = 0;
	while(sys.time < t_end)
	{
		if(sys.getTimeIndex()%outputInterval==0)
		{
			stringstream ss;
			//cout << "Frame: " << fileIndex << endl;
			ss << data_folder << "/" << fileIndex << ".txt";
			sys.writeToFile(ss.str());
			fileIndex++;
		}
		sys.DoTimeStep();
		ofile << sys.time                 << ", "
		      << sys.stepTime             << ", "
		      << sys.stepNewtonIterations << ", "
		      << sys.stepKrylovIterations << ", "
		      << sys.precUpdated          << " ,     ";
		for (size_t i = 0; i < sys.stepNewtonIterations; ++i)
			ofile << sys.spikeSolveTime[i] << ", " << sys.spikeNumIter[i] << ",     ";
		ofile << endl;
	}
	printf("Total time to simulate: %f [s]\n",sys.timeToSimulate);
	ofile.close();

	return 0;
}

