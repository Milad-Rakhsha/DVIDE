#include "hip/hip_runtime.h"
#include "include.cuh"
#include "System.cuh"
#include "Element.cuh"
#include "Node.cuh"
#include "Particle.cuh"

bool updateDraw = 1;
bool showSphere = 1;

// Create the system (placed outside of main so it is available to the OpenGL code)
System sys;

#ifdef WITH_GLUT
OpenGLCamera oglcamera(camreal3(-1,1,-1),camreal3(0,0,0),camreal3(0,1,0),.01);

// OPENGL RENDERING CODE //
void changeSize(int w, int h) {
	if(h == 0) {h = 1;}
	float ratio = 1.0* w / h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, w, h);
	gluPerspective(45,ratio,.1,1000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(0.0,0.0,0.0,		0.0,0.0,-7,		0.0f,1.0f,0.0f);
}

void initScene(){
	GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };
	glClearColor (1.0, 1.0, 1.0, 0.0);
	glShadeModel (GL_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable (GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glHint (GL_POINT_SMOOTH_HINT, GL_DONT_CARE);
}

void drawAll()
{
	if(updateDraw){
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glEnable(GL_DEPTH_TEST);
		glFrontFace(GL_CCW);
		glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
		glDepthFunc(GL_LEQUAL);
		glClearDepth(1.0);

		glPointSize(2);
		glLoadIdentity();

		oglcamera.Update();

		for (int i = 0; i < sys.particles.size(); i++) {
			glColor3f(0.0f, 1.0f, 0.0f);
			glPushMatrix();
			float3 pos = sys.getXYZPositionParticle(i);
			glTranslatef(pos.x, pos.y, pos.z);
			glutSolidSphere(sys.particles[i].getRadius(), 30, 30);
			glPopMatrix();

			//indicate velocity
			glLineWidth(sys.elements[i].getRadius()*500);
			glColor3f(1.0f,0.0f,0.0f);
			glBegin(GL_LINES);
			glVertex3f(pos.x,pos.y,pos.z);
			float3 vel = sys.getXYZVelocityParticle(i);
			//cout << "v:" << vel.x << " " << vel.y << " " << vel.z << endl;
			pos +=2*sys.particles[i].getRadius()*normalize(vel);
			glVertex3f(pos.x,pos.y,pos.z);
			glEnd();
			glFlush();
		}

		for(int i=0;i<sys.elements.size();i++)
		{
			int xiDiv = sys.numContactPoints;

			double xiInc = 1/(static_cast<double>(xiDiv-1));

			if(showSphere)
			{
				glColor3f(0.0f,0.0f,1.0f);
				for(int j=0;j<xiDiv;j++)
				{
					glPushMatrix();
					float3 position = sys.getXYZPosition(i,xiInc*j);
					glTranslatef(position.x,position.y,position.z);
					glutSolidSphere(sys.elements[i].getRadius(),10,10);
					glPopMatrix();
				}
			}
			else
			{
				int xiDiv = sys.numContactPoints;
				double xiInc = 1/(static_cast<double>(xiDiv-1));
				glLineWidth(sys.elements[i].getRadius()*500);
				glColor3f(0.0f,1.0f,0.0f);
				glBegin(GL_LINE_STRIP);
				for(int j=0;j<sys.numContactPoints;j++)
				{
					float3 position = sys.getXYZPosition(i,xiInc*j);
					glVertex3f(position.x,position.y,position.z);
				}
				glEnd();
				glFlush();
			}
		}

		glutSwapBuffers();
	}
}

void renderSceneAll(){
	if(OGL){
		//if(sys.timeIndex%10==0)
			drawAll();
		sys.DoTimeStep();
	}
}

void CallBackKeyboardFunc(unsigned char key, int x, int y) {
	switch (key) {
	case 'w':
		oglcamera.Forward();
		break;
	case 's':
		oglcamera.Back();
		break;

	case 'd':
		oglcamera.Right();
		break;

	case 'a':
		oglcamera.Left();
		break;

	case 'q':
		oglcamera.Up();
		break;

	case 'e':
		oglcamera.Down();
		break;
	}
}

void CallBackMouseFunc(int button, int state, int x, int y) {
	oglcamera.SetPos(button, state, x, y);
}
void CallBackMotionFunc(int x, int y) {
	oglcamera.Move2D(x, y);
}
#endif
// END OPENGL RENDERING CODE //

int main(int argc, char** argv)
{
	// command line arguments
	// ImplicitBeamsGPU <numPartitions> <numBeamsPerSide> <solverType> <usePreconditioning> <elasticModulus> <dataFolder>
	// solverType: (0) BiCGStab, (1) BiCGStab1, (2) BiCGStab2, (3) MinRes

#ifdef WITH_GLUT
	bool visualize = true;
#endif

	sys.setTimeStep(1e-3, 1e-10);
	sys.setMaxNewtonIterations(20);
	sys.setMaxKrylovIterations(5000);
	sys.setNumPartitions((int)atoi(argv[1]));
	sys.numContactPoints = 30;


	double t_end = 5.0;
	int    precUpdateInterval = -1;
	float  precMaxKrylov = -1;
	int    outputInterval = 100;

	string data_folder;

//	if(argc == 3)
//	{
//		sys.setAlpha_HHT(-10);
//		int numElements = 1;
//		double length = 2;
//		double lengthElement = length/numElements;
//		double r = 0.01;
//		double E = 2e7;
//		double rho = 7810;
//		double nu = .3;
//		double P = -60;
//		Element element = Element(Node(0, 0, 0, 1, 0, 0), Node(lengthElement, 0, 0, 1, 0, 0), r, nu, E, rho);
//		sys.addElement(&element);
//		sys.addConstraint_AbsoluteFixed(0);
//		sys.numContactPoints = 10;
//
//		for(int i=1;i<numElements;i++)
//		{
//			element = Element(Node(i*lengthElement, 0, 0, 1, 0, 0), Node((i+1)*lengthElement, 0, 0, 1, 0, 0), r, nu, E, rho);
//			sys.addElement(&element);
//			sys.addConstraint_RelativeFixed(sys.elements[i-1], 1,sys.elements[i], 0);
//		}
//		sys.addForce(&element,1,make_float3(0,P,0));
//
////		// should get deflection = PL^3/(3EI)
////		double I = .25*PI*r*r*r*r;
////		double deflection = P*pow(length,3)/(3*E*I);
////		cout << deflection << endl;
////		cin.get();
//	}
//	else
	{
		sys.fullJacobian = 1;
		double length = 1;
		double r = .02;
		double E = 2e11;
		double rho = 2200;
		double nu = .3;
		int numElementsPerSide = atoi(argv[2]);
		sys.setSolverType((int)atoi(argv[3]));
		sys.setPrecondType(atoi(argv[4]));
		if(atoi(argv[4])) {
			sys.preconditionerUpdateModulus = precUpdateInterval;
			sys.preconditionerMaxKrylovIterations = precMaxKrylov;
		}
		E = atof(argv[5]);
		data_folder = argv[6];

		Element element;
		int k = 0;
		// Add elements in x-direction
		for (int j = 0; j < numElementsPerSide+1; j++) {
			for (int i = 0; i < numElementsPerSide; i++) {
				element = Element(Node(i*length, 0, j*length, 1, 0, 0),
								  Node((i+1)*length, 0, j*length, 1, 0, 0),
								  r, nu, E, rho);
				sys.addElement(&element);
				k++;
				if(k%100==0) printf("Elements %d\n",k);
			}
		}

		// Add elements in z-direction
		for (int j = 0; j < numElementsPerSide+1; j++) {
			for (int i = 0; i < numElementsPerSide; i++) {
				element = Element(Node(j*length, 0, i*length, 0, 0, 1),
								  Node(j*length, 0, (i+1)*length, 0, 0, 1),
								  r, nu, E, rho);
				sys.addElement(&element);
				k++;
				if(k%100==0) printf("Elements %d\n",k);
			}
		}

		// Fix corners to ground
		sys.addConstraint_AbsoluteSpherical(sys.elements[0], 0);
		sys.addConstraint_AbsoluteSpherical(sys.elements[2*numElementsPerSide*(numElementsPerSide+1)-numElementsPerSide], 0);
		sys.addConstraint_AbsoluteSpherical(sys.elements[numElementsPerSide*(numElementsPerSide+1)-numElementsPerSide], 0);
		sys.addConstraint_AbsoluteSpherical(sys.elements[2*numElementsPerSide*(numElementsPerSide+1)-1], 1);
		sys.addConstraint_AbsoluteSpherical(sys.elements[numElementsPerSide*(numElementsPerSide+1)-1], 1);


		// Constrain x-strands together
		for(int j=0; j < numElementsPerSide+1; j++)
		{
			for(int i=0; i < numElementsPerSide-1; i++)
			{
				sys.addConstraint_RelativeFixed(
						sys.elements[i+j*numElementsPerSide], 1,
						sys.elements[i+1+j*numElementsPerSide], 0);
			}
		}

		// Constrain z-strands together
		int offset = numElementsPerSide*(numElementsPerSide+1);
		for(int j=0; j < numElementsPerSide+1; j++)
		{
			for(int i=0; i < numElementsPerSide-1; i++)
			{
				sys.addConstraint_RelativeFixed(
						sys.elements[i+offset+j*numElementsPerSide], 1,
						sys.elements[i+offset+1+j*numElementsPerSide], 0);
			}
		}

		// Constrain cross-streams together
		for(int j=0; j < numElementsPerSide; j++)
		{
			for(int i=0; i < numElementsPerSide; i++)
			{
				sys.addConstraint_RelativeSpherical(
						sys.elements[i*numElementsPerSide+j], 0,
						sys.elements[offset+i+j*numElementsPerSide], 0);
			}
		}

		for(int i=0; i < numElementsPerSide; i++)
		{
			sys.addConstraint_RelativeSpherical(
						sys.elements[numElementsPerSide-1+numElementsPerSide*i], 1,
						sys.elements[2*offset-numElementsPerSide+i], 0);
		}

		for(int i=0; i < numElementsPerSide; i++)
		{
			sys.addConstraint_RelativeSpherical(
						sys.elements[numElementsPerSide*(numElementsPerSide+1)+numElementsPerSide-1+numElementsPerSide*i], 1,
						sys.elements[numElementsPerSide*numElementsPerSide+i], 0);
		}
	}

	printf("%d, %d, %d\n",sys.elements.size(),sys.constraints.size(),12*sys.elements.size()+sys.constraints.size());
	sys.initializeSystem();
	printf("System initialized!\n");
	sys.printSolverParams();
	
#ifdef WITH_GLUT
	if(visualize)
	{
		glutInit(&argc, argv);
		glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
		glutInitWindowPosition(0,0);
		glutInitWindowSize(1024	,512);
		glutCreateWindow("MAIN");
		glutDisplayFunc(renderSceneAll);
		glutIdleFunc(renderSceneAll);
		glutReshapeFunc(changeSize);
		glutIgnoreKeyRepeat(0);
		glutKeyboardFunc(CallBackKeyboardFunc);
		glutMouseFunc(CallBackMouseFunc);
		glutMotionFunc(CallBackMotionFunc);
		initScene();
		glutMainLoop();
	}
#endif

	stringstream ss_m;
	ss_m << data_folder << "/" << "timing_" << atoi(argv[1]) << "_" << atoi(argv[2]) << "_" << atoi(argv[3]) << "_" << atoi(argv[4]) << "_" << atof(argv[5]) << ".txt";
	string timing_file_name = ss_m.str();
	ofstream ofile(timing_file_name.c_str());
	
	// if you don't want to visualize, then output the data
	int fileIndex = 0;
	while(sys.time < t_end)
	{
		if(sys.getTimeIndex()%outputInterval==0)
		{
			stringstream ss;
			//cout << "Frame: " << fileIndex << endl;
			ss << data_folder << "/" << fileIndex << ".txt";
			sys.writeToFile(ss.str());
			fileIndex++;
		}
		sys.DoTimeStep();
		ofile << sys.time                 << ", "
		      << sys.stepTime             << ", "
		      << sys.stepNewtonIterations << ", "
		      << sys.stepKrylovIterations << ", "
		      << sys.precUpdated          << " ,     ";
		for (size_t i = 0; i < sys.stepNewtonIterations; ++i)
			ofile << sys.spikeSolveTime[i] << ", " << sys.spikeNumIter[i] << ",     ";
		ofile << endl;
	}
	printf("Total time to simulate: %f [s]\n",sys.timeToSimulate);
	ofile.close();

	return 0;
}

